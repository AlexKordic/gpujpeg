#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_encoder.h"
#include "gpujpeg_util.h"

#define WARPS_NUM 8


#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
/** Allocate huffman tables in constant memory */
__constant__ struct gpujpeg_table_huffman_encoder gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT];
/** Pass huffman tables to encoder */
extern struct gpujpeg_table_huffman_encoder (*gpujpeg_encoder_table_huffman)[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT] = &gpujpeg_huffman_gpu_encoder_table_huffman;
#endif

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_encoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];

/**
 * Write marker to compressed data
 * 
 * @param data_compressed  Data compressed
 * @oaran marker  Marker to write (JPEG_MARKER_...)
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_marker(data_compressed, marker) { \
    *data_compressed = 0xFF;\
    data_compressed++; \
    *data_compressed = (uint8_t)(marker); \
    data_compressed++; }


/**
 * Adds up to 32 bits at once.
 * Codeword value must be aligned to left (most significant bits).
 */
__device__ inline void 
gpujpeg_huffman_gpu_encoder_emit_bits(unsigned int & remaining_bits, int & byte_count, int & bit_count, uint8_t * const out_ptr, const unsigned int packed_code_word) {
    // decompose packed codeword into the msb-aligned value and bit-length of the value
    const unsigned int code_word = packed_code_word & ~31;
    const unsigned int code_bit_size = packed_code_word & 31;
    
    // concatenate with remaining bits
    remaining_bits |= code_word >> bit_count;
    bit_count += code_bit_size;
    if (bit_count >= 8) {
        do {
            const unsigned int out_byte = remaining_bits >> 24;
            out_ptr[byte_count++] = out_byte;
            if(0xff == out_byte) {
                out_ptr[byte_count++] = 0;
            }
            
            remaining_bits <<= 8;
            bit_count -= 8;
        } while (bit_count >= 8);
        
        remaining_bits = code_word << (code_bit_size - bit_count);
        remaining_bits &= 0xfffffffe << (31 - bit_count);
    }
}


__device__ static void
gpujpeg_huffman_gpu_encode_value(unsigned int & out_nbits, unsigned int & out_cword, const int preceding_zero_count, const int value,
                                 const struct gpujpeg_table_huffman_encoder * const d_table) {
    out_cword = value;
    int absolute = value;
    if ( value < 0 ) {
        // valu eis now absolute value of input
        absolute = -absolute;
        // For a negative input, want temp2 = bitwise complement of abs(input)
        // This code assumes we are on a two's complement machine
        out_cword--;
    }

    // Find the number of bits needed for the magnitude of the coefficient
    out_nbits = 0;
    while ( absolute ) {
        out_nbits++;
        absolute >>= 1;
    }
    
    // trim remaining bits
    out_cword &= (1 << out_nbits) - 1;
    
    // find prefix of the codeword and size of the prefix
    const int prefix_idx = preceding_zero_count * 16 + out_nbits;
    out_cword |= d_table->code[prefix_idx] << out_nbits;
    out_nbits += d_table->size[prefix_idx];
}


__device__ void
gpujpeg_huffman_gpu_encoder_flush_codewords(unsigned int * const s_out, unsigned int * &data_compressed, int & remaining_codewords, const int tid) {
    // this works for up to 4 * 32 remaining codewords
    if(remaining_codewords) {
        // pad remianing codewords with extra zero-sized codewords, not to have to use special case in serialization kernel, which saves 4 codewords at once
        s_out[remaining_codewords + tid] = 0;
        
        // save all remaining codewords at once (together with some zero sized padding codewords)
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        
        // update codeword counter
        data_compressed += remaining_codewords;
        remaining_codewords = 0;
    }
}


/**
 * Encode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ int
gpujpeg_huffman_gpu_encoder_encode_block(int16_t * block, unsigned int * &data_compressed, unsigned int * const s_out,
                int & remaining_codewords, int *last_dc, int tid,
                struct gpujpeg_table_huffman_encoder* d_table_dc, struct gpujpeg_table_huffman_encoder* d_table_ac)
{
    // each thread loads a pair of values (pair after zigzag reordering)
    const int load_idx = tid * 2;
    int in_even = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx]];
    const int in_odd = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx + 1]];
    
    // compute number of zeros preceding the thread's even value
    const unsigned int even_nonzero_bitmap = 1 | __ballot(in_even); // DC coefficient is always treated as nonzero
    const unsigned int odd_nonzero_bitmap = __ballot(in_odd);
    const unsigned int nonzero_mask = (1 << tid) - 1;
    const int even_nonzero_count = __clz(even_nonzero_bitmap & nonzero_mask);
    const int odd_nonzero_count = __clz(odd_nonzero_bitmap & nonzero_mask);
    const int zeros_before_even = (min(odd_nonzero_count, even_nonzero_count) + tid - 32) * 2
                                + (odd_nonzero_count > even_nonzero_count ? 1 : 0);
    
    
    // true if any nonzero pixel follows thread's even pixel
    const unsigned int follow_mask = ~(nonzero_mask >> 1);
    const bool nonzero_follows = follow_mask & (even_nonzero_bitmap | odd_nonzero_bitmap);
    
    // count of consecutive zeros before odd value (either one more than 
    // even if even is zero or none if even value itself is nonzero)
    const int zeros_before_odd = in_even || !tid ? 0 : zeros_before_even + 1;
    
    // pointer to LUT for encoding thread's even value 
    // (only thread #0 uses DC table, others use AC table)
    const struct gpujpeg_table_huffman_encoder * d_table_even = d_table_ac;
    
    // first thread handles special DC coefficient
    if(0 == tid) {
        // first thread uses DC table for its even value
        d_table_even = d_table_dc;
        
        // update last DC coefficient
        const int original_in_even = in_even;
        in_even -= *last_dc;
        *last_dc = original_in_even;
    }
    
    // each thread gets codeword for its two pixels
    unsigned int even_code_size = 0, even_code_value = 0, odd_code_size = 0, odd_code_value = 0;
    if(nonzero_follows || !tid) {
        gpujpeg_huffman_gpu_encode_value(even_code_size, even_code_value, zeros_before_even & 0xf, in_even, d_table_even);
        gpujpeg_huffman_gpu_encode_value(odd_code_size, odd_code_value, zeros_before_odd & 0xf, in_odd, d_table_ac);
    }
    
    // last thread writes "end of block" value if last coefficient is zero
    if(tid == 31 && !in_odd) {
        odd_code_size = d_table_ac->size[256];
        odd_code_value = d_table_ac->code[256];
    }
    
    // concatenate both codewords into one if they are short enough
    if(even_code_size + odd_code_size < 27) {
        even_code_value = (even_code_value << odd_code_size) | odd_code_value;
        even_code_size += odd_code_size;
        odd_code_size = 0;
        odd_code_value = 0;
    }
    
    // each thread get number of preceding nonzero codewords and total number of nonzero codewords in this block
    const unsigned int even_codeword_presence = __ballot(even_code_size);
    const unsigned int odd_codeword_presence = __ballot(odd_code_size);
    const int codeword_offset = __popc(nonzero_mask & even_codeword_presence)
                              + __popc(nonzero_mask & odd_codeword_presence);
    
    // each thread saves its values into temporary shared buffer
    if(even_code_size) {
        s_out[remaining_codewords + codeword_offset] = even_code_size + (even_code_value << (32 - even_code_size));
        if(odd_code_size) {
            s_out[remaining_codewords + codeword_offset + 1] = odd_code_size + (odd_code_value << (32 - odd_code_size));
        }
    }
    
    // advance count of codewords in shared memory buffer
    remaining_codewords += __popc(odd_codeword_presence) + __popc(even_codeword_presence);
    
    // flush some codewords to global memory if there are too many of them in shared buffer
    const int flush_count = 32 * 4; // = half of the buffer
    if(remaining_codewords > flush_count) {
        // move first half of the buffer into output buffer in global memory and update output pointer
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        data_compressed += flush_count;
        
        // shift remaining codewords to begin of the buffer and update their count
        ((uint4*)s_out)[tid] = ((uint4*)s_out)[flush_count / 4 + tid];  // 4 for 4 uints in uint4
        remaining_codewords -= flush_count;
    }
        
    // nothing to fail here
    return 0;
}




/**
 * Huffman encoder kernel
 * 
 * @return void
 */
__global__ void
gpujpeg_huffman_encoder_encode_kernel(
    struct gpujpeg_component* d_component,
    struct gpujpeg_segment* d_segment,
    int comp_count,
    int segment_count, 
    uint8_t* d_data_compressed
#ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    ,struct gpujpeg_table_huffman_encoder* d_table_y_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_y_ac
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac
#endif
)
{    
#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    // Get huffman tables from constant memory
    struct gpujpeg_table_huffman_encoder* d_table_y_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_y_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC];
#endif
    
    int warpidx = threadIdx.x >> 5;
    int tid = threadIdx.x & 31;

    __shared__ uint4 s_out_all[64 * WARPS_NUM];
    unsigned int * s_out = (unsigned int*)(s_out_all + warpidx * 64);
    
    // Number of remaining codewords in shared buffer
    int remaining_codewords = 0;
    
    // Select Segment
    int segment_index = blockIdx.x * WARPS_NUM + warpidx;
    if ( segment_index >= segment_count )
        return;
    
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    
    // Initialize huffman coder
    int dc[GPUJPEG_MAX_COMPONENT_COUNT];
    for ( int comp = 0; comp < GPUJPEG_MAX_COMPONENT_COUNT; comp++ )
        dc[comp] = 0;
    
    // Prepare data pointers
    unsigned int * data_compressed = (unsigned int*)(d_data_compressed + segment->data_compressed_index);
    unsigned int * data_compressed_start = data_compressed;
    
    // Non-interleaving mode
    if ( comp_count == 1 ) {

        // Get component for current scan
        struct gpujpeg_component* component = &d_component[segment->scan_index];

        // Get component data for MCU (first block)
        int16_t* block = &component->d_data_quantized[(segment_index * component->segment_mcu_count) * component->mcu_size];

        // Get coder parameters
        int & last_dc = dc[segment->scan_index];

        // Get huffman tables
        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
            d_table_dc = d_table_y_dc;
            d_table_ac = d_table_y_ac;
        } else {
            d_table_dc = d_table_cbcr_dc;
            d_table_ac = d_table_cbcr_ac;
        }
            
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            // Encode 8x8 block
            if (gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac) != 0)
                break;
            block += component->mcu_size;
        }
    }
    // Interleaving mode
    else {
        int segment_index = segment->scan_segment_index; //TODO asi nepotrebne
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            //assert(segment->scan_index == 0);
            for ( int comp = 0; comp < comp_count; comp++ ) {
                struct gpujpeg_component* component = &d_component[comp];

                // Prepare mcu indexes
                int mcu_index_x = (segment_index * component->segment_mcu_count + mcu_index) % component->mcu_count_x;
                int mcu_index_y = (segment_index * component->segment_mcu_count + mcu_index) / component->mcu_count_x;
                // Compute base data index
                int data_index_base = mcu_index_y * (component->mcu_size * component->mcu_count_x) + mcu_index_x * (component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                
                // For all vertical 8x8 blocks
                for ( int y = 0; y < component->sampling_factor.vertical; y++ ) {
                    // Compute base row data index
                    int data_index_row = data_index_base + y * (component->mcu_count_x * component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                    // For all horizontal 8x8 blocks
                    for ( int x = 0; x < component->sampling_factor.horizontal; x++ ) {
                        // Compute 8x8 block data index
                        int data_index = data_index_row + x * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
                        
                        // Get component data for MCU
                        int16_t* block = &component->d_data_quantized[data_index];
                        
                        // Get coder parameters
                        int & last_dc = dc[comp];
            
                        // Get huffman tables
                        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
                        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
                        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
                            d_table_dc = d_table_y_dc;
                            d_table_ac = d_table_y_ac;
                        } else {
                            d_table_dc = d_table_cbcr_dc;
                            d_table_ac = d_table_cbcr_ac;
                        }
                        
                        // Encode 8x8 block
                        gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac);
                    }
                }
            }
        }
    }

    // flush remaining codewords
    gpujpeg_huffman_gpu_encoder_flush_codewords(s_out, data_compressed, remaining_codewords, tid);
    
    // Set number of codewords.
    if (tid == 0 ) {
        segment->data_compressed_size = data_compressed - data_compressed_start;
    }
    __syncthreads();
}



#define SERIALIZATION_THREADS_PER_TBLOCK 192


/**
 * Codeword serialization kernel.
 * 
 * @return void
 */
__global__ static void
gpujpeg_huffman_encoder_serialization_kernel(
    struct gpujpeg_segment* d_segment,
    int segment_count, 
    uint8_t* d_data_compressed
) {    
    // Temp buffer for all threads of the threadblock
    __shared__ uint4 s_temp_all[2 * SERIALIZATION_THREADS_PER_TBLOCK];

    // Thread's 32 bytes in shared memory for output composition
    uint4 * const s_temp = s_temp_all + threadIdx.x * 2;
    
    // Select Segment
    int segment_index = blockIdx.x * SERIALIZATION_THREADS_PER_TBLOCK + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    // Thread's segment
    struct gpujpeg_segment* const segment = &d_segment[segment_index];
    
    // Input and output pointers
    uint4 * const d_dest_stream_start = (uint4*)(d_data_compressed + segment->data_compressed_index);
    uint4 * d_dest_stream = d_dest_stream_start;
    const uint4 * d_src_codewords = d_dest_stream_start;
    
    // number of bytes in the temp buffer, remaining bits and their count
    int byte_count = 0, bit_count = 0;
    unsigned int remaining_bits = 0;
    
    // "data_compressed_size" is now initialize dto number of codewords to be serialized
    const int cword_count = segment->data_compressed_size;
    for( int cword_idx = 0; cword_idx < cword_count; cword_idx += 4 ) // reading 4 codewords at once
    {
        // read 4 codewords and advance input pointer to next ones
        const uint4 cwords = *(d_src_codewords++);
        
        // encode all 4 codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.x);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.y);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.z);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.w);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count > 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
    }
    
    // Emit left bits
    gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, 0xfe000007);

    // Terminate codestream with restart marker
    ((uint8_t*)s_temp)[byte_count + 0] = 0xFF;
    ((uint8_t*)s_temp)[byte_count + 1] = GPUJPEG_MARKER_RST0 + (segment->scan_segment_index % 8);
    
    // flush remaining bytes
    d_dest_stream[0] = s_temp[0];
    d_dest_stream[1] = s_temp[1];
    
    // Set compressed size
    segment->data_compressed_size = (d_dest_stream - d_dest_stream_start) * 16 + byte_count + 2;
}




/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_init()
{
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(
        (const char*)gpujpeg_huffman_gpu_encoder_order_natural,
        gpujpeg_order_natural, 
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init");
    
    return 0;
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_encode(struct gpujpeg_encoder* encoder)
{    
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    assert(coder->param.restart_interval > 0);
    
    int comp_count = 1;
    if ( coder->param.interleaved == 1 )
        comp_count = coder->param_image.comp_count;
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);

    // Configure more shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_encode_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_serialization_kernel), hipFuncCachePreferShared);
            
    // Run encoder kernel
    dim3 thread(32 * WARPS_NUM);
    dim3 grid(gpujpeg_div_and_round_up(coder->segment_count, (thread.x / 32)));
    gpujpeg_huffman_encoder_encode_kernel<<<grid, thread>>>(
        coder->d_component, 
        coder->d_segment, 
        comp_count,
        coder->segment_count, 
        coder->d_data_compressed
    #ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    #endif
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman encoding failed");
    
    
    // Run codeword serialization kernel
    const int num_serialization_tblocks = gpujpeg_div_and_round_up(coder->segment_count, SERIALIZATION_THREADS_PER_TBLOCK);
    gpujpeg_huffman_encoder_serialization_kernel<<<num_serialization_tblocks, SERIALIZATION_THREADS_PER_TBLOCK>>>(
        coder->d_segment, 
        coder->segment_count, 
        coder->d_data_compressed
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Codeword serialization failed");
    
    
    return 0;
}
