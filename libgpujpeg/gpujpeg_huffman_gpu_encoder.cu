#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_encoder.h"
#include "gpujpeg_util.h"

#define WARPS_NUM 8



/**
 * Huffman coding tables in constant memory - each has 257 items (256 + 1 extra)
 * There are are 4 of them - one after another, in following order:
 *    - luminance (Y) AC
 *    - luminance (Y) DC
 *    - chroma (cb/cr) AC
 *    - chroma (cb/cr) DC
 */
__device__ uint32_t gpujpeg_huffman_gpu_lut[(256 + 1) * 4];

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_encoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];

/** Value decomposition in constant memory (input range from -4096 to 4095  ... both inclusive) */
__device__ unsigned int gpujpeg_huffman_value_decomposition[8 * 1024];

/** Size of occupied part of output buffer */
__device__ unsigned int gpujpeg_huffman_output_byte_count;

/**
 * Write marker to compressed data
 * 
 * @param data_compressed  Data compressed
 * @oaran marker  Marker to write (JPEG_MARKER_...)
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_marker(data_compressed, marker) { \
    *data_compressed = 0xFF;\
    data_compressed++; \
    *data_compressed = (uint8_t)(marker); \
    data_compressed++; }


/** Initializes coefficient decomposition table in global memory */
static __global__ void
gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel() {
    // fetch some value
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int value = tid - 4096;
    
    // decompose it
    unsigned int value_code = value;
    int absolute = value;
    if ( value < 0 ) {
        // valu eis now absolute value of input
        absolute = -absolute;
        // For a negative input, want temp2 = bitwise complement of abs(input)
        // This code assumes we are on a two's complement machine
        value_code--;
    }

    // Find the number of bits needed for the magnitude of the coefficient
    unsigned int value_nbits = 0;
    while ( absolute ) {
        value_nbits++;
        absolute >>= 1;
    }
    
    // save result packed into unsigned int (value bits are left aligned in MSBs and size is right aligned in LSBs)
    gpujpeg_huffman_value_decomposition[tid] = value_nbits | (value_code << (32 - value_nbits));
//     printf("%+04d: %08x\n", value, gpujpeg_huffman_value_decomposition[tid]);
}


/**
 * Adds up to 32 bits at once.
 * Codeword value must be aligned to left (most significant bits).
 */
__device__ inline void 
gpujpeg_huffman_gpu_encoder_emit_bits(unsigned int & remaining_bits, int & byte_count, int & bit_count, uint8_t * const out_ptr, const unsigned int packed_code_word)
{
    // decompose packed codeword into the msb-aligned value and bit-length of the value
    const unsigned int code_word = packed_code_word & ~31;
    const unsigned int code_bit_size = packed_code_word & 31;
    
    // concatenate with remaining bits
    remaining_bits |= code_word >> bit_count;
    bit_count += code_bit_size;
    if (bit_count >= 8) {
        do {
            const unsigned int out_byte = remaining_bits >> 24;
            out_ptr[byte_count++] = out_byte;
            if(0xff == out_byte) {
                out_ptr[byte_count++] = 0;
            }
            
            remaining_bits <<= 8;
            bit_count -= 8;
        } while (bit_count >= 8);
        
        remaining_bits = code_word << (code_bit_size - bit_count);
        remaining_bits &= 0xfffffffe << (31 - bit_count);
    }
}


__device__ static unsigned int
gpujpeg_huffman_gpu_encode_value(const int preceding_zero_count, const int coefficient,
                                 const int huffman_lut_offset)
{
    // value bits are in MSBs (left aligned) and bit size of the value is in LSBs (right aligned)
    const unsigned int packed_value = gpujpeg_huffman_value_decomposition[4096 + coefficient];
    
    // decompose value info into upshifted value and value's bit size
    const int value_nbits = packed_value & 0xf;
    const unsigned int value_code = packed_value & ~0xf;
    
    // find prefix of the codeword and size of the prefix
    const int huffman_lut_idx = huffman_lut_offset + preceding_zero_count * 16 + value_nbits;
    const unsigned int packed_prefix = gpujpeg_huffman_gpu_lut[huffman_lut_idx];
    const unsigned int prefix_nbits = packed_prefix & 31;
    
    // compose packed codeword with its size
    return (packed_prefix + value_nbits) | (value_code >> prefix_nbits);
}


__device__ void
gpujpeg_huffman_gpu_encoder_flush_codewords(unsigned int * const s_out, unsigned int * &data_compressed, int & remaining_codewords, const int tid) {
    // this works for up to 4 * 32 remaining codewords
    if(remaining_codewords) {
        // pad remianing codewords with extra zero-sized codewords, not to have to use special case in serialization kernel, which saves 4 codewords at once
        s_out[remaining_codewords + tid] = 0;
        
        // save all remaining codewords at once (together with some zero sized padding codewords)
        *((uint4*)data_compressed) = ((uint4*)s_out)[tid];
        
        // update codeword counter
        data_compressed += remaining_codewords;
        remaining_codewords = 0;
    }
}


/**
 * Encode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ int
gpujpeg_huffman_gpu_encoder_encode_block(const int16_t * block, unsigned int * &data_compressed, unsigned int * const s_out,
                int & remaining_codewords, const int last_dc_idx, int tid, const int huffman_lut_offset)
{
    // each thread loads a pair of values (pair after zigzag reordering)
    const int load_idx = tid * 2;
    int in_even = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx]];
    const int in_odd = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx + 1]];
    
    // compute preceding zero count for even coefficient (actually compute the count multiplied by 16)
    const unsigned int nonzero_mask = (1 << tid) - 1;
    const unsigned int nonzero_bitmap_0 = 1 | __ballot(in_even);  // DC is always treated as nonzero
    const unsigned int nonzero_bitmap_1 = __ballot(in_odd);
    const unsigned int nonzero_bitmap_pairs = nonzero_bitmap_0 | nonzero_bitmap_1;
    
    const int zero_pair_count = __clz(nonzero_bitmap_pairs & nonzero_mask);
    int zeros_before_even = 2 * (zero_pair_count + tid - 32);
    if((0x80000000 >> zero_pair_count) > (nonzero_bitmap_1 & nonzero_mask)) {
        zeros_before_even += 1;
    }
    
    // true if any nonzero pixel follows thread's odd pixel
    const bool nonzero_follows = nonzero_bitmap_pairs & ~nonzero_mask;
    
    // count of consecutive zeros before odd value (either one more than 
    // even if even is zero or none if even value itself is nonzero)
    // (the count is actually multiplied by 16)
    int zeros_before_odd = in_even || !tid ? 0 : zeros_before_even + 1;
    
    // clear zero counts if no nonzero pixel follows (so that no 16-zero symbols will be emited)
    // otherwise only trim extra bits from the counts of following zeros
    const int zero_count_mask = nonzero_follows ? 0xF : 0;
    zeros_before_even &= zero_count_mask;
    zeros_before_odd &= zero_count_mask;
    
    // pointer to LUT for encoding thread's even value 
    // (only thread #0 uses DC table, others use AC table)
    int even_lut_offset = huffman_lut_offset;
    
    // first thread handles special DC coefficient
    if(0 == tid) {
        // first thread uses DC part of the table for its even value
        even_lut_offset += 256 + 1;
        
        // update last DC coefficient (saved at the special place at the end of the shared bufer)
        const int original_in_even = in_even;
        in_even -= ((int*)s_out)[last_dc_idx];
        ((int*)s_out)[last_dc_idx] = original_in_even;
    }
    
    // last thread handles special block-termination symbol
    if(0 == ((tid ^ 31) | in_odd)) {
        // this causes selection of huffman symbol at index 256 (which contains the termination symbol)
        zeros_before_odd = 16;
    }
    
    // each thread gets codeword for its two pixels
    unsigned int even_code = gpujpeg_huffman_gpu_encode_value(zeros_before_even, in_even, even_lut_offset);
    unsigned int odd_code = gpujpeg_huffman_gpu_encode_value(zeros_before_odd, in_odd, huffman_lut_offset);
            
    // concatenate both codewords into one if they are short enough
    const unsigned int even_code_size = even_code & 31;
    const unsigned int odd_code_size = odd_code & 31;
    const unsigned int total_size = even_code_size + odd_code_size;
    if(total_size <= 27) {
        even_code = total_size | ((odd_code & ~31) >> even_code_size) | (even_code & ~31);
        odd_code = 0;
    }
    
    // each thread get number of preceding nonzero codewords and total number of nonzero codewords in this block
    const unsigned int even_codeword_presence = __ballot(even_code);
    const unsigned int odd_codeword_presence = __ballot(odd_code);
    const int codeword_offset = __popc(nonzero_mask & even_codeword_presence)
                              + __popc(nonzero_mask & odd_codeword_presence);
    
    // each thread saves its values into temporary shared buffer
    if(even_code) {
        s_out[remaining_codewords + codeword_offset] = even_code;
        if(odd_code) {
            s_out[remaining_codewords + codeword_offset + 1] = odd_code;
        }
    }
    
    // advance count of codewords in shared memory buffer
    remaining_codewords += __popc(odd_codeword_presence) + __popc(even_codeword_presence);
    
    // flush some codewords to global memory if there are too many of them in shared buffer
    const int flush_count = 32 * 4; // = half of the buffer
    if(remaining_codewords > flush_count) {
        // move first half of the buffer into output buffer in global memory and update output pointer
        *((uint4*)data_compressed) = ((uint4*)s_out)[tid];
        data_compressed += flush_count;
        
        // shift remaining codewords to begin of the buffer and update their count
        ((uint4*)s_out)[tid] = ((uint4*)s_out)[flush_count / 4 + tid];  // 4 for 4 uints in uint4
        remaining_codewords -= flush_count;
    }
        
    // nothing to fail here
    return 0;
}




/**
 * Huffman encoder kernel
 * 
 * @return void
 */
template <bool CONTINUOUS_BLOCK_LIST>
__launch_bounds__(WARPS_NUM * 32, 1024 / (WARPS_NUM * 32))
__global__ void
gpujpeg_huffman_encoder_encode_kernel(
    struct gpujpeg_segment* d_segment,
    int segment_count, 
    uint8_t* d_data_compressed,
    const uint64_t* const d_block_list,
    int16_t* const d_data_quantized,
    struct gpujpeg_component* const d_component,
    const int comp_count
) {    
    int warpidx = threadIdx.x >> 5;
    int tid = threadIdx.x & 31;

    __shared__ uint4 s_out_all[(64 + 1) * WARPS_NUM];
    unsigned int * s_out = (unsigned int*)(s_out_all + warpidx * (64 + 1));
    
    // Number of remaining codewords in shared buffer
    int remaining_codewords = 0;
    
    // Select Segment
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    const int segment_index = block_idx * WARPS_NUM + warpidx;
    
    // fires thread initializes compact output size for next kernel
    if(0 == tid && 0 == warpidx && 0 == block_idx) {
        gpujpeg_huffman_output_byte_count = 0;
    }
    
    // stop if out of segment bounds
    if ( segment_index >= segment_count )
        return;
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    
    // Initialize last DC coefficients
    if(tid < 3) {
        s_out[256 + tid] = 0;
    }
    
    // Prepare data pointers
    unsigned int * data_compressed = (unsigned int*)(d_data_compressed + segment->data_temp_index);
    unsigned int * data_compressed_start = data_compressed;
    
    // Pre-add thread ID to output pointer (it's allways used only with it)
    data_compressed += (tid * 4);
    
    // Encode all block in segment
    if(CONTINUOUS_BLOCK_LIST) {
        // Get component for current scan
        const struct gpujpeg_component* component = &d_component[segment->scan_index];
        
        // mcu size of the component
        const int comp_mcu_size = component->mcu_size;
        
        // Get component data for MCU (first block)
        const int16_t* block = component->d_data_quantized + (segment->scan_segment_index * component->segment_mcu_count) * comp_mcu_size;
        
        // Get huffman table offset
        const int huffman_table_offset = component->type == GPUJPEG_COMPONENT_LUMINANCE ? 0 : (256 + 1) * 2; // possibly skips luminance tables
        
        // Encode MCUs in segment
        for (int block_count = segment->mcu_count; block_count--;) {
            // Encode 8x8 block
            gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, 256, tid, huffman_table_offset);
            
            // Advance to next block
            block += comp_mcu_size;
        }
    } else {
        // Pointer to segment's list of 8x8 blocks and their count
        const uint64_t* packed_block_info_ptr = d_block_list + segment->block_index_list_begin;
        
        // Encode all blocks
        for(int block_count = segment->block_count; block_count--;) {
            // Get pointer to next block input data and info about its color type
            const uint64_t packed_block_info = *(packed_block_info_ptr++);
            
            // Get coder parameters
            const int last_dc_idx = 256 + (packed_block_info & 0x7f);
            
            // Get offset to right part of huffman table
            const int huffman_table_offset = packed_block_info & 0x80 ? (256 + 1) * 2 : 0; // possibly skips luminance tables
            
            // Source data pointer
            int16_t* block = &d_data_quantized[packed_block_info >> 8];
                        
            // Encode 8x8 block
            gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, last_dc_idx, tid, huffman_table_offset);
        }
    }

    // flush remaining codewords
    gpujpeg_huffman_gpu_encoder_flush_codewords(s_out, data_compressed, remaining_codewords, tid);
    
    // Set number of codewords.
    if (tid == 0 ) {
        segment->data_compressed_size = data_compressed - data_compressed_start;
    }
}



#define SERIALIZATION_THREADS_PER_TBLOCK 192


/**
 * Codeword serialization kernel.
 * 
 * @return void
 */
__launch_bounds__(SERIALIZATION_THREADS_PER_TBLOCK, 1536 / SERIALIZATION_THREADS_PER_TBLOCK)
__global__ static void
gpujpeg_huffman_encoder_serialization_kernel(
    struct gpujpeg_segment* d_segment,
    int segment_count, 
    uint8_t* d_data_compressed
) {    
    // Temp buffer for all threads of the threadblock
    __shared__ uint4 s_temp_all[2 * SERIALIZATION_THREADS_PER_TBLOCK];

    // Thread's 32 bytes in shared memory for output composition
    uint4 * const s_temp = s_temp_all + threadIdx.x * 2;
    
    // Select Segment
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int segment_index = block_idx * SERIALIZATION_THREADS_PER_TBLOCK + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    // Thread's segment
    struct gpujpeg_segment* const segment = &d_segment[segment_index];
    
    // Input and output pointers
    uint4 * const d_dest_stream_start = (uint4*)(d_data_compressed + segment->data_temp_index);
    uint4 * d_dest_stream = d_dest_stream_start;
    const uint4 * d_src_codewords = d_dest_stream_start;
    
    // number of bytes in the temp buffer, remaining bits and their count
    int byte_count = 0, bit_count = 0;
    unsigned int remaining_bits = 0;
    
    // "data_compressed_size" is now initialized to number of codewords to be serialized
    for(int cword_tuple_count = (segment->data_compressed_size + 3) >> 2; cword_tuple_count--; ) // reading 4 codewords at once
    {
        // read 4 codewords and advance input pointer to next ones
        const uint4 cwords = *(d_src_codewords++);
        
        // encode first pair of codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.x);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.y);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count >= 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
        
        // encode other two codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.z);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.w);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count >= 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
    }
    
    // Emit left bits
    gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, 0xfe000007);

    // Terminate codestream with restart marker
    ((uint8_t*)s_temp)[byte_count + 0] = 0xFF;
    ((uint8_t*)s_temp)[byte_count + 1] = GPUJPEG_MARKER_RST0 + (segment->scan_segment_index % 8);
    
    // flush remaining bytes
    d_dest_stream[0] = s_temp[0];
    d_dest_stream[1] = s_temp[1];
    
    // Set compressed size
    segment->data_compressed_size = (d_dest_stream - d_dest_stream_start) * 16 + byte_count + 2;
}




/**
 * Huffman coder output compaction kernel.
 * 
 * @return void
 */
__global__ static void
gpujpeg_huffman_encoder_compaction_kernel (
    struct gpujpeg_segment* const d_segment,
    const int segment_count, 
    const uint8_t* const d_src,
    uint8_t* const d_dest
) {    
    // get some segment (size of threadblocks is 32 x N, so threadIdx.y is warp index)
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    const int segment_idx = threadIdx.y + block_idx * blockDim.y;
    if(segment_idx >= segment_count) {
        return;
    }
    
    // temp variables for all warps
    __shared__ uint4* volatile s_out_ptrs[WARPS_NUM];
    
    // get info about the segment
    const unsigned int segment_byte_count = (d_segment[segment_idx].data_compressed_size + 15) & ~15;  // number of bytes rounded up to multiple of 16
    const unsigned int segment_in_offset = d_segment[segment_idx].data_temp_index;  // this should be aligned at least to 16byte boundary
    
    // first thread of each warp reserves space in output buffer
    if(0 == threadIdx.x) {
         const unsigned int segment_out_offset = atomicAdd(&gpujpeg_huffman_output_byte_count, segment_byte_count);
         s_out_ptrs[threadIdx.y] = (uint4*)(d_dest + segment_out_offset);
         d_segment[segment_idx].data_compressed_index = segment_out_offset;
    }
    
    // all threads read output buffer offset for their segment and prepare input and output pointers and number of copy iterations
    const uint4 * d_in = threadIdx.x + (uint4*)(d_src + segment_in_offset);
    uint4 * d_out = threadIdx.x + s_out_ptrs[threadIdx.y];
    unsigned int copy_iterations = segment_byte_count / 512; // 512 is number of bytes copied in each iteration (32 threads * 16 bytes per thread)
    
    // copy the data!
    while(copy_iterations--) {
        *d_out = *d_in;
        d_out += 32;
        d_in += 32;
    }
    
    // copy remaining bytes (less than 512 bytes)
    if((threadIdx.x * 16) < (segment_byte_count & 511)) {
        *d_out = *d_in;
    }
}


/** Adds packed coefficients into the GPU version of Huffman lookup table. */
void
gpujpeg_huffman_gpu_add_packed_table(uint32_t * const dest, const struct gpujpeg_table_huffman_encoder * const src, const bool is_ac) {
    // make a upshifted copy of the table for GPU encoding
    for ( int i = 0; i <= 256; i++ ) {
        const int size = src->size[i & 0xFF];
        dest[i] = (src->code[i & 0xFF] << (32 - size)) | size;
    }
    
    // reserve first index in GPU version of AC table for special purposes
    if ( is_ac ) {
        dest[0] = 0;
    }
}



/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_init(const struct gpujpeg_encoder * encoder)
{
    
    // Initialize decomposition lookup table
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel), hipFuncCachePreferShared);
    gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel<<<32, 256>>>();  // 8192 threads total
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Decomposition LUT initialization failed");
    
    // compose GPU version of the huffman LUT and copy it into GPU mempry
    uint32_t gpujpeg_huffman_cpu_lut[(256 + 1) * 4];
    gpujpeg_huffman_gpu_add_packed_table(gpujpeg_huffman_cpu_lut + 257 * 0, &encoder->table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC], true);
    gpujpeg_huffman_gpu_add_packed_table(gpujpeg_huffman_cpu_lut + 257 * 1, &encoder->table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC], false);
    gpujpeg_huffman_gpu_add_packed_table(gpujpeg_huffman_cpu_lut + 257 * 2, &encoder->table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC], true);
    gpujpeg_huffman_gpu_add_packed_table(gpujpeg_huffman_cpu_lut + 257 * 3, &encoder->table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC], false);
    hipMemcpyToSymbol(HIP_SYMBOL(
        gpujpeg_huffman_gpu_lut),
        gpujpeg_huffman_cpu_lut,
        (256 + 1) * 4 * sizeof(*gpujpeg_huffman_gpu_lut),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init (Huffman LUT copy)");
    
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(
        (const char*)gpujpeg_huffman_gpu_encoder_order_natural,
        gpujpeg_order_natural, 
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init (natural order copy)");
    
    return 0;
}


dim3
gpujpeg_huffman_gpu_encoder_grid_size(int tblock_count)
{
    dim3 size(tblock_count);
    while(size.x > 0xffff) {
        size.x = (size.x + 1) >> 1;
        size.y <<= 1;
    }
    return size;
}




/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_encode(struct gpujpeg_encoder* encoder, unsigned int * output_byte_count)
{   
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    assert(coder->param.restart_interval > 0);
    
    // Configure more shared memory for all kernels
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_encode_kernel<true>), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_encode_kernel<false>), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_serialization_kernel), hipFuncCachePreferShared);
    
    // Select encoder kernel which either expects continuos segments of blocks or uses block lists
    int comp_count = 1;
    if ( coder->param.interleaved == 1 )
        comp_count = coder->param_image.comp_count;
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);
    
    // Run encoder kernel
    dim3 thread(32 * WARPS_NUM);
    dim3 grid = gpujpeg_huffman_gpu_encoder_grid_size(gpujpeg_div_and_round_up(coder->segment_count, (thread.x / 32)));
    if(comp_count == 1) {
        gpujpeg_huffman_encoder_encode_kernel<true><<<grid, thread>>>(
            coder->d_segment, 
            coder->segment_count, 
            coder->d_temp_huffman,
            coder->d_block_list,
            coder->d_data_quantized,
            coder->d_component,
            comp_count
        );
    } else { 
        gpujpeg_huffman_encoder_encode_kernel<false><<<grid, thread>>>(
            coder->d_segment, 
            coder->segment_count, 
            coder->d_temp_huffman,
            coder->d_block_list,
            coder->d_data_quantized,
            coder->d_component,
            comp_count
        );
    }
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman encoding failed");
    
    // Run codeword serialization kernel
    const int num_serialization_tblocks = gpujpeg_div_and_round_up(coder->segment_count, SERIALIZATION_THREADS_PER_TBLOCK);
    const dim3 serialization_grid = gpujpeg_huffman_gpu_encoder_grid_size(num_serialization_tblocks);
    gpujpeg_huffman_encoder_serialization_kernel<<<num_serialization_tblocks, SERIALIZATION_THREADS_PER_TBLOCK>>>(
        coder->d_segment, 
        coder->segment_count, 
        coder->d_temp_huffman
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Codeword serialization failed");
    
    // Run output compaction kernel (one warp per segment)
    const dim3 compaction_thread(32, WARPS_NUM);
    const dim3 compaction_grid = gpujpeg_huffman_gpu_encoder_grid_size(gpujpeg_div_and_round_up(coder->segment_count, WARPS_NUM));
    gpujpeg_huffman_encoder_compaction_kernel<<<compaction_grid, compaction_thread>>>(
        coder->d_segment,
        coder->segment_count,
        coder->d_temp_huffman,
        coder->d_data_compressed
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman output compaction failed");
    
    // Read and return number of occupied bytes
    hipMemcpyFromSymbol(output_byte_count, HIP_SYMBOL(gpujpeg_huffman_output_byte_count), sizeof(unsigned int), 0, hipMemcpyDeviceToHost);
    gpujpeg_cuda_check_error("Huffman output size getting failed");
    
    return 0;
}
