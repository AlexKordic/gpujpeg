#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_encoder.h"
#include "gpujpeg_util.h"

#define WARPS_NUM 8


#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
/** Allocate huffman tables in constant memory */
__constant__ struct gpujpeg_table_huffman_encoder gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT];
/** Pass huffman tables to encoder */
extern struct gpujpeg_table_huffman_encoder (*gpujpeg_encoder_table_huffman)[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT] = &gpujpeg_huffman_gpu_encoder_table_huffman;
#endif

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_encoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];

/** Value decomposition in constant memory (input range from -4096 to 4095  ... both inclusive) */
__device__ unsigned int gpujpeg_huffman_value_decomposition[8 * 1024];

/**
 * Write marker to compressed data
 * 
 * @param data_compressed  Data compressed
 * @oaran marker  Marker to write (JPEG_MARKER_...)
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_marker(data_compressed, marker) { \
    *data_compressed = 0xFF;\
    data_compressed++; \
    *data_compressed = (uint8_t)(marker); \
    data_compressed++; }


/** Initializes coefficient decomposition table in global memory */
static __global__ void
gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel() {
    // fetch some value
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int value = tid - 4096;
    
    // decompose it
    unsigned int value_code = value;
    int absolute = value;
    if ( value < 0 ) {
        // valu eis now absolute value of input
        absolute = -absolute;
        // For a negative input, want temp2 = bitwise complement of abs(input)
        // This code assumes we are on a two's complement machine
        value_code--;
    }

    // Find the number of bits needed for the magnitude of the coefficient
    unsigned int value_nbits = 0;
    while ( absolute ) {
        value_nbits++;
        absolute >>= 1;
    }
    
    // save result packed into unsigned int (value bits are left aligned in MSBs and size is right aligned in LSBs)
    gpujpeg_huffman_value_decomposition[tid] = value_nbits | (value_code << (32 - value_nbits));
//     printf("%+04d: %08x\n", value, gpujpeg_huffman_value_decomposition[tid]);
}


/**
 * Adds up to 32 bits at once.
 * Codeword value must be aligned to left (most significant bits).
 */
__device__ inline void 
gpujpeg_huffman_gpu_encoder_emit_bits(unsigned int & remaining_bits, int & byte_count, int & bit_count, uint8_t * const out_ptr, const unsigned int packed_code_word)
{
    // decompose packed codeword into the msb-aligned value and bit-length of the value
    const unsigned int code_word = packed_code_word & ~31;
    const unsigned int code_bit_size = packed_code_word & 31;
    
    // concatenate with remaining bits
    remaining_bits |= code_word >> bit_count;
    bit_count += code_bit_size;
    if (bit_count >= 8) {
        do {
            const unsigned int out_byte = remaining_bits >> 24;
            out_ptr[byte_count++] = out_byte;
            if(0xff == out_byte) {
                out_ptr[byte_count++] = 0;
            }
            
            remaining_bits <<= 8;
            bit_count -= 8;
        } while (bit_count >= 8);
        
        remaining_bits = code_word << (code_bit_size - bit_count);
        remaining_bits &= 0xfffffffe << (31 - bit_count);
    }
}


__device__ static unsigned int
gpujpeg_huffman_gpu_encode_value(const int preceding_zero_count, const int value,
                                 const struct gpujpeg_table_huffman_encoder * const d_table)
{
    // value bits are in MSBs (left aligned) and bit size of the value is in LSBs (right aligned)
    const unsigned int packed_value = gpujpeg_huffman_value_decomposition[4096 + value];
    
    // decompose value info into upshifted value and value's bit size
    const int value_nbits = packed_value & 0xf;
    const unsigned int value_code = packed_value & ~0xf;
    
    // find prefix of the codeword and size of the prefix
    const int prefix_idx = (preceding_zero_count & 0xf) * 16 + value_nbits;
    const unsigned int codeword = d_table->code[prefix_idx];
    const unsigned int prefix_nbits = codeword & 31;
    
    // compose packed codeword with its size
    return (codeword + value_nbits) | (value_code >> prefix_nbits);
}


__device__ void
gpujpeg_huffman_gpu_encoder_flush_codewords(unsigned int * const s_out, unsigned int * &data_compressed, int & remaining_codewords, const int tid) {
    // this works for up to 4 * 32 remaining codewords
    if(remaining_codewords) {
        // pad remianing codewords with extra zero-sized codewords, not to have to use special case in serialization kernel, which saves 4 codewords at once
        s_out[remaining_codewords + tid] = 0;
        
        // save all remaining codewords at once (together with some zero sized padding codewords)
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        
        // update codeword counter
        data_compressed += remaining_codewords;
        remaining_codewords = 0;
    }
}


/**
 * Encode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ int
gpujpeg_huffman_gpu_encoder_encode_block(int16_t * block, unsigned int * &data_compressed, unsigned int * const s_out,
                int & remaining_codewords, int *last_dc, int tid,
                struct gpujpeg_table_huffman_encoder* d_table_dc, struct gpujpeg_table_huffman_encoder* d_table_ac)
{
    // each thread loads a pair of values (pair after zigzag reordering)
    const int load_idx = tid * 2;
    int in_even = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx]];
    const int in_odd = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx + 1]];
    
    // compute number of zeros preceding the thread's even value
    const unsigned int even_nonzero_bitmap = 1 | __ballot(in_even); // DC coefficient is always treated as nonzero
    const unsigned int odd_nonzero_bitmap = __ballot(in_odd);
    const unsigned int nonzero_mask = (1 << tid) - 1;
    const int even_nonzero_count = __clz(even_nonzero_bitmap & nonzero_mask);
    const int odd_nonzero_count = __clz(odd_nonzero_bitmap & nonzero_mask);
    const int zeros_before_even = (min(odd_nonzero_count, even_nonzero_count) + tid - 32) * 2
                                + (odd_nonzero_count > even_nonzero_count ? 1 : 0);
    
    
    // true if any nonzero pixel follows thread's even pixel
    const unsigned int follow_mask = ~(nonzero_mask >> 1);
    const bool nonzero_follows = follow_mask & (even_nonzero_bitmap | odd_nonzero_bitmap);
    
    // count of consecutive zeros before odd value (either one more than 
    // even if even is zero or none if even value itself is nonzero)
    const int zeros_before_odd = in_even || !tid ? 0 : zeros_before_even + 1;
    
    // pointer to LUT for encoding thread's even value 
    // (only thread #0 uses DC table, others use AC table)
    const struct gpujpeg_table_huffman_encoder * d_table_even = d_table_ac;
    
    // first thread handles special DC coefficient
    if(0 == tid) {
        // first thread uses DC table for its even value
        d_table_even = d_table_dc;
        
        // update last DC coefficient
        const int original_in_even = in_even;
        in_even -= *last_dc;
        *last_dc = original_in_even;
    }
    
    // each thread gets codeword for its two pixels
    unsigned int even_code = 0, odd_code = 0;
    if(nonzero_follows || !tid) {
        even_code = gpujpeg_huffman_gpu_encode_value(zeros_before_even, in_even, d_table_even);
        odd_code = gpujpeg_huffman_gpu_encode_value(zeros_before_odd, in_odd, d_table_ac);
    }
    
    // last thread writes "end of block" value if last coefficient is zero
    if(tid == 31 && !in_odd) {
        odd_code = d_table_ac->code[256];
    }
    
    // concatenate both codewords into one if they are short enough
    const unsigned int even_code_size = even_code & 31;
    const unsigned int odd_code_size = odd_code & 31;
    const unsigned int total_size = even_code_size + odd_code_size;
    if(total_size <= 27) {
        even_code = total_size | ((odd_code & ~31) >> even_code_size) | (even_code & ~31);
        odd_code = 0;
    }
    
    // each thread get number of preceding nonzero codewords and total number of nonzero codewords in this block
    const unsigned int even_codeword_presence = __ballot(even_code);
    const unsigned int odd_codeword_presence = __ballot(odd_code);
    const int codeword_offset = __popc(nonzero_mask & even_codeword_presence)
                              + __popc(nonzero_mask & odd_codeword_presence);
    
    // each thread saves its values into temporary shared buffer
    if(even_code) {
        s_out[remaining_codewords + codeword_offset] = even_code;
        if(odd_code) {
            s_out[remaining_codewords + codeword_offset + 1] = odd_code;
        }
    }
    
    // advance count of codewords in shared memory buffer
    remaining_codewords += __popc(odd_codeword_presence) + __popc(even_codeword_presence);
    
    // flush some codewords to global memory if there are too many of them in shared buffer
    const int flush_count = 32 * 4; // = half of the buffer
    if(remaining_codewords > flush_count) {
        // move first half of the buffer into output buffer in global memory and update output pointer
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        data_compressed += flush_count;
        
        // shift remaining codewords to begin of the buffer and update their count
        ((uint4*)s_out)[tid] = ((uint4*)s_out)[flush_count / 4 + tid];  // 4 for 4 uints in uint4
        remaining_codewords -= flush_count;
    }
        
    // nothing to fail here
    return 0;
}




/**
 * Huffman encoder kernel
 * 
 * @return void
 */
__global__ void
gpujpeg_huffman_encoder_encode_kernel(
    struct gpujpeg_component* d_component,
    struct gpujpeg_segment* d_segment,
    int comp_count,
    int segment_count, 
    uint8_t* d_data_compressed
#ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    ,struct gpujpeg_table_huffman_encoder* d_table_y_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_y_ac
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac
#endif
)
{    
#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    // Get huffman tables from constant memory
    struct gpujpeg_table_huffman_encoder* d_table_y_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_y_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC];
#endif
    
    int warpidx = threadIdx.x >> 5;
    int tid = threadIdx.x & 31;

    __shared__ uint4 s_out_all[64 * WARPS_NUM];
    unsigned int * s_out = (unsigned int*)(s_out_all + warpidx * 64);
    
    // Number of remaining codewords in shared buffer
    int remaining_codewords = 0;
    
    // Select Segment
    int segment_index = blockIdx.x * WARPS_NUM + warpidx;
    if ( segment_index >= segment_count )
        return;
    
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    
    // Initialize huffman coder
    int dc[GPUJPEG_MAX_COMPONENT_COUNT];
    for ( int comp = 0; comp < GPUJPEG_MAX_COMPONENT_COUNT; comp++ )
        dc[comp] = 0;
    
    // Prepare data pointers
    unsigned int * data_compressed = (unsigned int*)(d_data_compressed + segment->data_compressed_index);
    unsigned int * data_compressed_start = data_compressed;
    
    // Non-interleaving mode
    if ( comp_count == 1 ) {

        // Get component for current scan
        struct gpujpeg_component* component = &d_component[segment->scan_index];

        // Get component data for MCU (first block)
        int16_t* block = &component->d_data_quantized[(segment_index * component->segment_mcu_count) * component->mcu_size];

        // Get coder parameters
        int & last_dc = dc[segment->scan_index];

        // Get huffman tables
        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
            d_table_dc = d_table_y_dc;
            d_table_ac = d_table_y_ac;
        } else {
            d_table_dc = d_table_cbcr_dc;
            d_table_ac = d_table_cbcr_ac;
        }
            
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            // Encode 8x8 block
            if (gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac) != 0)
                break;
            block += component->mcu_size;
        }
    }
    // Interleaving mode
    else {
        int segment_index = segment->scan_segment_index; //TODO asi nepotrebne
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            //assert(segment->scan_index == 0);
            for ( int comp = 0; comp < comp_count; comp++ ) {
                struct gpujpeg_component* component = &d_component[comp];

                // Prepare mcu indexes
                int mcu_index_x = (segment_index * component->segment_mcu_count + mcu_index) % component->mcu_count_x;
                int mcu_index_y = (segment_index * component->segment_mcu_count + mcu_index) / component->mcu_count_x;
                // Compute base data index
                int data_index_base = mcu_index_y * (component->mcu_size * component->mcu_count_x) + mcu_index_x * (component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                
                // For all vertical 8x8 blocks
                for ( int y = 0; y < component->sampling_factor.vertical; y++ ) {
                    // Compute base row data index
                    int data_index_row = data_index_base + y * (component->mcu_count_x * component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                    // For all horizontal 8x8 blocks
                    for ( int x = 0; x < component->sampling_factor.horizontal; x++ ) {
                        // Compute 8x8 block data index
                        int data_index = data_index_row + x * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
                        
                        // Get component data for MCU
                        int16_t* block = &component->d_data_quantized[data_index];
                        
                        // Get coder parameters
                        int & last_dc = dc[comp];
            
                        // Get huffman tables
                        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
                        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
                        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
                            d_table_dc = d_table_y_dc;
                            d_table_ac = d_table_y_ac;
                        } else {
                            d_table_dc = d_table_cbcr_dc;
                            d_table_ac = d_table_cbcr_ac;
                        }
                        
                        // Encode 8x8 block
                        gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac);
                    }
                }
            }
        }
    }

    // flush remaining codewords
    gpujpeg_huffman_gpu_encoder_flush_codewords(s_out, data_compressed, remaining_codewords, tid);
    
    // Set number of codewords.
    if (tid == 0 ) {
        segment->data_compressed_size = data_compressed - data_compressed_start;
    }
    __syncthreads();
}



#define SERIALIZATION_THREADS_PER_TBLOCK 192


/**
 * Codeword serialization kernel.
 * 
 * @return void
 */
__global__ static void
gpujpeg_huffman_encoder_serialization_kernel(
    struct gpujpeg_segment* d_segment,
    int segment_count, 
    uint8_t* d_data_compressed
) {    
    // Temp buffer for all threads of the threadblock
    __shared__ uint4 s_temp_all[2 * SERIALIZATION_THREADS_PER_TBLOCK];

    // Thread's 32 bytes in shared memory for output composition
    uint4 * const s_temp = s_temp_all + threadIdx.x * 2;
    
    // Select Segment
    int segment_index = blockIdx.x * SERIALIZATION_THREADS_PER_TBLOCK + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    // Thread's segment
    struct gpujpeg_segment* const segment = &d_segment[segment_index];
    
    // Input and output pointers
    uint4 * const d_dest_stream_start = (uint4*)(d_data_compressed + segment->data_compressed_index);
    uint4 * d_dest_stream = d_dest_stream_start;
    const uint4 * d_src_codewords = d_dest_stream_start;
    
    // number of bytes in the temp buffer, remaining bits and their count
    int byte_count = 0, bit_count = 0;
    unsigned int remaining_bits = 0;
    
    // "data_compressed_size" is now initialize dto number of codewords to be serialized
    const int cword_count = segment->data_compressed_size;
    for( int cword_idx = 0; cword_idx < cword_count; cword_idx += 4 ) // reading 4 codewords at once
    {
        // read 4 codewords and advance input pointer to next ones
        const uint4 cwords = *(d_src_codewords++);
        
        // encode all 4 codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.x);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.y);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.z);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.w);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count > 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
    }
    
    // Emit left bits
    gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, 0xfe000007);

    // Terminate codestream with restart marker
    ((uint8_t*)s_temp)[byte_count + 0] = 0xFF;
    ((uint8_t*)s_temp)[byte_count + 1] = GPUJPEG_MARKER_RST0 + (segment->scan_segment_index % 8);
    
    // flush remaining bytes
    d_dest_stream[0] = s_temp[0];
    d_dest_stream[1] = s_temp[1];
    
    // Set compressed size
    segment->data_compressed_size = (d_dest_stream - d_dest_stream_start) * 16 + byte_count + 2;
}




/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_init()
{
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(
        (const char*)gpujpeg_huffman_gpu_encoder_order_natural,
        gpujpeg_order_natural, 
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init");
    
    return 0;
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_encode(struct gpujpeg_encoder* encoder)
{    
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    assert(coder->param.restart_interval > 0);
    
    int comp_count = 1;
    if ( coder->param.interleaved == 1 )
        comp_count = coder->param_image.comp_count;
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);

    // Configure more shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_encode_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_serialization_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel), hipFuncCachePreferShared);
    
    // Initialize decomposition lookup table
    gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel<<<32, 256>>>();  // 8192 threads total
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Decomposition LUT initialization failed");
    
    // Run encoder kernel
    dim3 thread(32 * WARPS_NUM);
    dim3 grid(gpujpeg_div_and_round_up(coder->segment_count, (thread.x / 32)));
    gpujpeg_huffman_encoder_encode_kernel<<<grid, thread>>>(
        coder->d_component, 
        coder->d_segment, 
        comp_count,
        coder->segment_count, 
        coder->d_data_compressed
    #ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    #endif
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman encoding failed");
    
    
    // Run codeword serialization kernel
    const int num_serialization_tblocks = gpujpeg_div_and_round_up(coder->segment_count, SERIALIZATION_THREADS_PER_TBLOCK);
    gpujpeg_huffman_encoder_serialization_kernel<<<num_serialization_tblocks, SERIALIZATION_THREADS_PER_TBLOCK>>>(
        coder->d_segment, 
        coder->segment_count, 
        coder->d_data_compressed
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Codeword serialization failed");
    
    
    return 0;
}
