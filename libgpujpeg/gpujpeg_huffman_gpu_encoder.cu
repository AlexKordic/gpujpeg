#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_encoder.h"
#include "gpujpeg_util.h"

#define WARPS_NUM 8


#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
/** Allocate huffman tables in constant memory */
__constant__ struct gpujpeg_table_huffman_encoder gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT];
/** Pass huffman tables to encoder */
extern struct gpujpeg_table_huffman_encoder (*gpujpeg_encoder_table_huffman)[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT] = &gpujpeg_huffman_gpu_encoder_table_huffman;
#endif

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_encoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];

/** Value decomposition in constant memory (input range from -4096 to 4095  ... both inclusive) */
__device__ unsigned int gpujpeg_huffman_value_decomposition[8 * 1024];

/** Size of occupied part of output buffer */
__device__ unsigned int gpujpeg_huffman_output_byte_count;

/**
 * Write marker to compressed data
 * 
 * @param data_compressed  Data compressed
 * @oaran marker  Marker to write (JPEG_MARKER_...)
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_marker(data_compressed, marker) { \
    *data_compressed = 0xFF;\
    data_compressed++; \
    *data_compressed = (uint8_t)(marker); \
    data_compressed++; }


/** Initializes coefficient decomposition table in global memory */
static __global__ void
gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel() {
    // fetch some value
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int value = tid - 4096;
    
    // decompose it
    unsigned int value_code = value;
    int absolute = value;
    if ( value < 0 ) {
        // valu eis now absolute value of input
        absolute = -absolute;
        // For a negative input, want temp2 = bitwise complement of abs(input)
        // This code assumes we are on a two's complement machine
        value_code--;
    }

    // Find the number of bits needed for the magnitude of the coefficient
    unsigned int value_nbits = 0;
    while ( absolute ) {
        value_nbits++;
        absolute >>= 1;
    }
    
    // save result packed into unsigned int (value bits are left aligned in MSBs and size is right aligned in LSBs)
    gpujpeg_huffman_value_decomposition[tid] = value_nbits | (value_code << (32 - value_nbits));
//     printf("%+04d: %08x\n", value, gpujpeg_huffman_value_decomposition[tid]);
    
    // first thread also initializes size of final output, not to have to do it in separate memcpy/kernel
    if(0 == tid) {
        gpujpeg_huffman_output_byte_count = 0;
    }
}


/**
 * Adds up to 32 bits at once.
 * Codeword value must be aligned to left (most significant bits).
 */
__device__ inline void 
gpujpeg_huffman_gpu_encoder_emit_bits(unsigned int & remaining_bits, int & byte_count, int & bit_count, uint8_t * const out_ptr, const unsigned int packed_code_word)
{
    // decompose packed codeword into the msb-aligned value and bit-length of the value
    const unsigned int code_word = packed_code_word & ~31;
    const unsigned int code_bit_size = packed_code_word & 31;
    
    // concatenate with remaining bits
    remaining_bits |= code_word >> bit_count;
    bit_count += code_bit_size;
    if (bit_count >= 8) {
        do {
            const unsigned int out_byte = remaining_bits >> 24;
            out_ptr[byte_count++] = out_byte;
            if(0xff == out_byte) {
                out_ptr[byte_count++] = 0;
            }
            
            remaining_bits <<= 8;
            bit_count -= 8;
        } while (bit_count >= 8);
        
        remaining_bits = code_word << (code_bit_size - bit_count);
        remaining_bits &= 0xfffffffe << (31 - bit_count);
    }
}


__device__ static unsigned int
gpujpeg_huffman_gpu_encode_value(const int preceding_zero_count_idx, const int value,
                                 const struct gpujpeg_table_huffman_encoder * const d_table)
{
    // value bits are in MSBs (left aligned) and bit size of the value is in LSBs (right aligned)
    const unsigned int packed_value = gpujpeg_huffman_value_decomposition[4096 + value];
    
    // decompose value info into upshifted value and value's bit size
    const int value_nbits = packed_value & 0xf;
    const unsigned int value_code = packed_value & ~0xf;
    
    // find prefix of the codeword and size of the prefix
    const unsigned int prefix_code = d_table->gcode[preceding_zero_count_idx | value_nbits];
    const unsigned int prefix_nbits = prefix_code & 31;
    
    // compose packed codeword with its size
    return (prefix_code + value_nbits) | (value_code >> prefix_nbits);
}


__device__ void
gpujpeg_huffman_gpu_encoder_flush_codewords(unsigned int * const s_out, unsigned int * &data_compressed, int & remaining_codewords, const int tid) {
    // this works for up to 4 * 32 remaining codewords
    if(remaining_codewords) {
        // pad remianing codewords with extra zero-sized codewords, not to have to use special case in serialization kernel, which saves 4 codewords at once
        s_out[remaining_codewords + tid] = 0;
        
        // save all remaining codewords at once (together with some zero sized padding codewords)
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        
        // update codeword counter
        data_compressed += remaining_codewords;
        remaining_codewords = 0;
    }
}


/**
 * Encode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ int
gpujpeg_huffman_gpu_encoder_encode_block(const int16_t * block, unsigned int * &data_compressed, unsigned int * const s_out,
                int & remaining_codewords, int *last_dc, int tid,
                struct gpujpeg_table_huffman_encoder* d_table_dc, struct gpujpeg_table_huffman_encoder* d_table_ac)
{
    // each thread loads a pair of values (pair after zigzag reordering)
    const int load_idx = tid * 2;
    int in_even = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx]];
    const int in_odd = block[gpujpeg_huffman_gpu_encoder_order_natural[load_idx + 1]];
    
    // compute preceding zero count for even coefficient (actually compute the count multiplied by 16)
    const unsigned int nonzero_mask = (1 << tid) - 1;
    const unsigned int nonzero_bitmap_0 = 1 | __ballot(in_even);  // DC is always treated as nonzero
    const unsigned int nonzero_bitmap_1 = __ballot(in_odd);
    const unsigned int nonzero_bitmap_pairs = nonzero_bitmap_0 | nonzero_bitmap_1;
    
    const int zero_pair_count = __clz(nonzero_bitmap_pairs & nonzero_mask);
    int zeros_before_even = 32 * (zero_pair_count + tid - 32);
    if((0x80000000 >> zero_pair_count) > (nonzero_bitmap_1 & nonzero_mask)) {
        zeros_before_even += 16;
    }
    
    // true if any nonzero pixel follows thread's odd pixel
    const bool nonzero_follows = nonzero_bitmap_pairs & ~nonzero_mask;
    
    // count of consecutive zeros before odd value (either one more than 
    // even if even is zero or none if even value itself is nonzero)
    // (the count is actually multiplied by 16)
    int zeros_before_odd = in_even || !tid ? 0 : zeros_before_even + 16;
    
    // clear zero counts if no nonzero pixel follows (so that no 16-zero symbols will be emited)
    // otherwise only trim extra bits from the counts of following zeros
    const int zero_count_mask = nonzero_follows ? 0xF0 : 0;
    zeros_before_even &= zero_count_mask;
    zeros_before_odd &= zero_count_mask;
    
    // pointer to LUT for encoding thread's even value 
    // (only thread #0 uses DC table, others use AC table)
    const struct gpujpeg_table_huffman_encoder * d_table_even = d_table_ac;
    
    // first thread handles special DC coefficient
    if(0 == tid) {
        // first thread uses DC table for its even value
        d_table_even = d_table_dc;
        
        // update last DC coefficient
        const int original_in_even = in_even;
        in_even -= *last_dc;
        *last_dc = original_in_even;
    }
    
    // last thread handles special block-termination symbol
    if(0 == ((tid ^ 31) | in_odd)) {
        // this causes selection of huffman symbol at index 256 (which contains the termination symbol)
        zeros_before_odd = 256;
    }
    
    // each thread gets codeword for its two pixels
    unsigned int even_code = gpujpeg_huffman_gpu_encode_value(zeros_before_even, in_even, d_table_even);
    unsigned int odd_code = gpujpeg_huffman_gpu_encode_value(zeros_before_odd, in_odd, d_table_ac);
    
    // concatenate both codewords into one if they are short enough
    const unsigned int even_code_size = even_code & 31;
    const unsigned int odd_code_size = odd_code & 31;
    const unsigned int total_size = even_code_size + odd_code_size;
    if(total_size <= 27) {
        even_code = total_size | ((odd_code & ~31) >> even_code_size) | (even_code & ~31);
        odd_code = 0;
    }
    
    // each thread get number of preceding nonzero codewords and total number of nonzero codewords in this block
    const unsigned int even_codeword_presence = __ballot(even_code);
    const unsigned int odd_codeword_presence = __ballot(odd_code);
    const int codeword_offset = __popc(nonzero_mask & even_codeword_presence)
                              + __popc(nonzero_mask & odd_codeword_presence);
    
    // each thread saves its values into temporary shared buffer
    if(even_code) {
        s_out[remaining_codewords + codeword_offset] = even_code;
        if(odd_code) {
            s_out[remaining_codewords + codeword_offset + 1] = odd_code;
        }
    }
    
    // advance count of codewords in shared memory buffer
    remaining_codewords += __popc(odd_codeword_presence) + __popc(even_codeword_presence);
    
    // flush some codewords to global memory if there are too many of them in shared buffer
    const int flush_count = 32 * 4; // = half of the buffer
    if(remaining_codewords > flush_count) {
        // move first half of the buffer into output buffer in global memory and update output pointer
        ((uint4*)data_compressed)[tid] = ((uint4*)s_out)[tid];
        data_compressed += flush_count;
        
        // shift remaining codewords to begin of the buffer and update their count
        ((uint4*)s_out)[tid] = ((uint4*)s_out)[flush_count / 4 + tid];  // 4 for 4 uints in uint4
        remaining_codewords -= flush_count;
    }
        
    // nothing to fail here
    return 0;
}




/**
 * Huffman encoder kernel
 * 
 * @return void
 */
__launch_bounds__(WARPS_NUM * 32, 1024 / (WARPS_NUM * 32))
__global__ void
gpujpeg_huffman_encoder_encode_kernel(
    struct gpujpeg_component* d_component,
    struct gpujpeg_segment* d_segment,
    int comp_count,
    int segment_count, 
    uint8_t* d_data_compressed
#ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    ,struct gpujpeg_table_huffman_encoder* d_table_y_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_y_ac
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac
#endif
)
{    
#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    // Get huffman tables from constant memory
    struct gpujpeg_table_huffman_encoder* d_table_y_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_y_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC];
#endif
    
    int warpidx = threadIdx.x >> 5;
    int tid = threadIdx.x & 31;

    __shared__ uint4 s_out_all[64 * WARPS_NUM];
    unsigned int * s_out = (unsigned int*)(s_out_all + warpidx * 64);
    
    // Number of remaining codewords in shared buffer
    int remaining_codewords = 0;
    
    // Select Segment
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    const int segment_index = block_idx * WARPS_NUM + warpidx;
    if ( segment_index >= segment_count )
        return;
    
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    const int segment_mcu_count = segment->mcu_count;
    
    // Initialize huffman coder
    int dc[GPUJPEG_MAX_COMPONENT_COUNT];
    for ( int comp = 0; comp < GPUJPEG_MAX_COMPONENT_COUNT; comp++ )
        dc[comp] = 0;
    
    // Prepare data pointers
    unsigned int * data_compressed = (unsigned int*)(d_data_compressed + segment->data_temp_index);
    unsigned int * data_compressed_start = data_compressed;
    
    // Non-interleaving mode
    if ( comp_count == 1 ) {
        int segment_index = segment->scan_segment_index;

        // Get component for current scan
        struct gpujpeg_component* component = &d_component[segment->scan_index];

        // Get component data for MCU (first block)
        const int16_t* block = &component->d_data_quantized[(segment_index * component->segment_mcu_count) * component->mcu_size];

        // Get coder parameters
        int & last_dc = dc[segment->scan_index];

        // Get huffman tables
        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
            d_table_dc = d_table_y_dc;
            d_table_ac = d_table_y_ac;
        } else {
            d_table_dc = d_table_cbcr_dc;
            d_table_ac = d_table_cbcr_ac;
        }
        
        // mcu size of the component
        const int comp_mcu_size = component->mcu_size;
            
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment_mcu_count; mcu_index++ ) {
            // Encode 8x8 block
            if (gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac) != 0)
                break;
            block += comp_mcu_size;
        }
    }
    // Interleaving mode
    else {
        int segment_index = segment->scan_segment_index; //TODO asi nepotrebne
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment_mcu_count; mcu_index++ ) {
            //assert(segment->scan_index == 0);
            for ( int comp = 0; comp < comp_count; comp++ ) {
                struct gpujpeg_component* component = &d_component[comp];

                // Prepare mcu indexes
                int mcu_index_x = (segment_index * component->segment_mcu_count + mcu_index) % component->mcu_count_x;
                int mcu_index_y = (segment_index * component->segment_mcu_count + mcu_index) / component->mcu_count_x;
                
                // Compute base data index
                int data_index_base = mcu_index_y * (component->mcu_size * component->mcu_count_x) + mcu_index_x * (component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                
                // Get huffman tables
                struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
                struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
                if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
                    d_table_dc = d_table_y_dc;
                    d_table_ac = d_table_y_ac;
                } else {
                    d_table_dc = d_table_cbcr_dc;
                    d_table_ac = d_table_cbcr_ac;
                }
                
                // Get sampling faactors
                const int comp_vertical_sampling = component->sampling_factor.vertical;
                const int comp_horizontal_sampling = component->sampling_factor.horizontal;
                
                // Pointer to quantized data of the component
                const int16_t* const d_component_data_quantized = component->d_data_quantized;
                
                // Size and count of MCUs in component
                const int comp_mcu_stride = component->mcu_count_x * component->mcu_size_x * GPUJPEG_BLOCK_SIZE;
                
                // For all vertical 8x8 blocks
                for ( int y = 0; y < comp_vertical_sampling; y++ ) {
                    // Compute base row data index
                    int data_index_row = data_index_base + y * comp_mcu_stride;
                    // For all horizontal 8x8 blocks
                    for ( int x = 0; x < comp_horizontal_sampling; x++ ) {
                        // Compute 8x8 block data index
                        int data_index = data_index_row + x * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
                        
                        // Get component data for MCU
                        const int16_t* block = d_component_data_quantized + data_index;
                        
                        // Get coder parameters
                        int & last_dc = dc[comp];
            
                        // Encode 8x8 block
                        gpujpeg_huffman_gpu_encoder_encode_block(block, data_compressed, s_out, remaining_codewords, &last_dc, tid, d_table_dc, d_table_ac);
                    }
                }
            }
        }
    }

    // flush remaining codewords
    gpujpeg_huffman_gpu_encoder_flush_codewords(s_out, data_compressed, remaining_codewords, tid);
    
    // Set number of codewords.
    if (tid == 0 ) {
        segment->data_compressed_size = data_compressed - data_compressed_start;
    }
    __syncthreads();
}



#define SERIALIZATION_THREADS_PER_TBLOCK 192


/**
 * Codeword serialization kernel.
 * 
 * @return void
 */
__launch_bounds__(SERIALIZATION_THREADS_PER_TBLOCK, 1536 / SERIALIZATION_THREADS_PER_TBLOCK)
__global__ static void
gpujpeg_huffman_encoder_serialization_kernel(
    struct gpujpeg_segment* d_segment,
    int segment_count, 
    uint8_t* d_data_compressed
) {    
    // Temp buffer for all threads of the threadblock
    __shared__ uint4 s_temp_all[2 * SERIALIZATION_THREADS_PER_TBLOCK];

    // Thread's 32 bytes in shared memory for output composition
    uint4 * const s_temp = s_temp_all + threadIdx.x * 2;
    
    // Select Segment
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int segment_index = block_idx * SERIALIZATION_THREADS_PER_TBLOCK + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    // Thread's segment
    struct gpujpeg_segment* const segment = &d_segment[segment_index];
    
    // Input and output pointers
    uint4 * const d_dest_stream_start = (uint4*)(d_data_compressed + segment->data_temp_index);
    uint4 * d_dest_stream = d_dest_stream_start;
    const uint4 * d_src_codewords = d_dest_stream_start;
    
    // number of bytes in the temp buffer, remaining bits and their count
    int byte_count = 0, bit_count = 0;
    unsigned int remaining_bits = 0;
    
    // "data_compressed_size" is now initialized to number of codewords to be serialized
    for(int cword_tuple_count = (segment->data_compressed_size + 3) >> 2; cword_tuple_count--; ) // reading 4 codewords at once
    {
        // read 4 codewords and advance input pointer to next ones
        const uint4 cwords = *(d_src_codewords++);
        
        // encode first pair of codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.x);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.y);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count >= 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
        
        // encode other two codewords
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.z);
        gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, cwords.w);
        
        // possibly flush output if have at least 16 bytes
        if(byte_count >= 16) {
            // write 16 bytes into destination buffer
            *(d_dest_stream++) = s_temp[0];
            
            // move remaining bytes to first half of the buffer
            s_temp[0] = s_temp[1];
            
            // update number of remaining bits
            byte_count -= 16;
        }
    }
    
    // Emit left bits
    gpujpeg_huffman_gpu_encoder_emit_bits(remaining_bits, byte_count, bit_count, (uint8_t*)s_temp, 0xfe000007);

    // Terminate codestream with restart marker
    ((uint8_t*)s_temp)[byte_count + 0] = 0xFF;
    ((uint8_t*)s_temp)[byte_count + 1] = GPUJPEG_MARKER_RST0 + (segment->scan_segment_index % 8);
    
    // flush remaining bytes
    d_dest_stream[0] = s_temp[0];
    d_dest_stream[1] = s_temp[1];
    
    // Set compressed size
    segment->data_compressed_size = (d_dest_stream - d_dest_stream_start) * 16 + byte_count + 2;
}




/**
 * Huffman coder output compaction kernel.
 * 
 * @return void
 */
__global__ static void
gpujpeg_huffman_encoder_compaction_kernel (
    struct gpujpeg_segment* const d_segment,
    const int segment_count, 
    const uint8_t* const d_src,
    uint8_t* const d_dest
) {    
    // get some segment (size of threadblocks is 32 x N, so threadIdx.y is warp index)
    const int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    const int segment_idx = threadIdx.y + block_idx * blockDim.y;
    if(segment_idx >= segment_count) {
        return;
    }
    
    // temp variables for all warps
    __shared__ volatile unsigned int s_out_offsets[WARPS_NUM];
    
    // get info about the segment
    const unsigned int segment_byte_count = (d_segment[segment_idx].data_compressed_size + 15) & ~15;  // number of bytes rounded up to multiple of 16
    const unsigned int segment_in_offset = d_segment[segment_idx].data_temp_index;  // this should be aligned at least to 16byte boundary
    
    // first thread of each warp reserves space in output buffer
    if(0 == threadIdx.x) {
         const unsigned int segment_out_offset = atomicAdd(&gpujpeg_huffman_output_byte_count, segment_byte_count);
         s_out_offsets[threadIdx.y] = segment_out_offset;
         d_segment[segment_idx].data_compressed_index = segment_out_offset;
    }
    
    // all threads read output buffer offset for their segment and prepare input and output pointers and number of copy iterations
    const uint4 * d_in = threadIdx.x + (uint4*)(d_src + segment_in_offset);
    uint4 * d_out = threadIdx.x + (uint4*)(d_dest + s_out_offsets[threadIdx.y]);
    unsigned int copy_iterations = segment_byte_count / 512; // 512 is number of bytes copied in each iteration (32 threads * 16 bytes per thread)
    
    // copy the data!
    while(copy_iterations--) {
        *d_out = *d_in;
        d_out += 32;
        d_in += 32;
    }
    
    // copy remaining bytes (less than 512 bytes)
    if((threadIdx.x * 16) < (segment_byte_count & 511)) {
        *d_out = *d_in;
    }
}



/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_init()
{
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(
        (const char*)gpujpeg_huffman_gpu_encoder_order_natural,
        gpujpeg_order_natural, 
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init");
    
    return 0;
}


dim3
gpujpeg_huffman_gpu_encoder_grid_size(int tblock_count)
{
    dim3 size(tblock_count);
    while(size.x > 0xffff) {
        size.x = (size.x + 1) >> 1;
        size.y <<= 1;
    }
    return size;
}




/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_encode(struct gpujpeg_encoder* encoder, unsigned int * output_byte_count)
{   
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    assert(coder->param.restart_interval > 0);
    
    int comp_count = 1;
    if ( coder->param.interleaved == 1 )
        comp_count = coder->param_image.comp_count;
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);

    // Configure more shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_encode_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_encoder_serialization_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel), hipFuncCachePreferShared);
    
    // Initialize decomposition lookup table
    gpujpeg_huffman_gpu_encoder_value_decomposition_init_kernel<<<32, 256>>>();  // 8192 threads total
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Decomposition LUT initialization failed");
    
    // Run encoder kernel
    dim3 thread(32 * WARPS_NUM);
    dim3 grid = gpujpeg_huffman_gpu_encoder_grid_size(gpujpeg_div_and_round_up(coder->segment_count, (thread.x / 32)));
    gpujpeg_huffman_encoder_encode_kernel<<<grid, thread>>>(
        coder->d_component, 
        coder->d_segment, 
        comp_count,
        coder->segment_count, 
        coder->d_temp_huffman
    #ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    #endif
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman encoding failed");
    
    
    // Run codeword serialization kernel
    const int num_serialization_tblocks = gpujpeg_div_and_round_up(coder->segment_count, SERIALIZATION_THREADS_PER_TBLOCK);
    const dim3 serialization_grid = gpujpeg_huffman_gpu_encoder_grid_size(num_serialization_tblocks);
    gpujpeg_huffman_encoder_serialization_kernel<<<num_serialization_tblocks, SERIALIZATION_THREADS_PER_TBLOCK>>>(
        coder->d_segment, 
        coder->segment_count, 
        coder->d_temp_huffman
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Codeword serialization failed");
    
    // Run output compaction kernel (one warp per segment)
    const dim3 compaction_thread(32, WARPS_NUM);
    const dim3 compaction_grid = gpujpeg_huffman_gpu_encoder_grid_size(gpujpeg_div_and_round_up(coder->segment_count, WARPS_NUM));
    gpujpeg_huffman_encoder_compaction_kernel<<<compaction_grid, compaction_thread>>>(
        coder->d_segment,
        coder->segment_count,
        coder->d_temp_huffman,
        coder->d_data_compressed
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman output compaction failed");
    
    // Read and return number of occupied bytes
    hipMemcpyFromSymbol(output_byte_count, HIP_SYMBOL(gpujpeg_huffman_output_byte_count), sizeof(unsigned int), 0, hipMemcpyDeviceToHost);
    gpujpeg_cuda_check_error("Huffman output size getting failed");
    
    return 0;
}
