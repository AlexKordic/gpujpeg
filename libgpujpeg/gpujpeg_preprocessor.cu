#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_preprocessor.h"
#include "gpujpeg_util.h"

/**
 * Clip [0,255] range
 */
inline __device__ float gpujpeg_clamp(float value)
{
    value = (value >= 0.0f) ? value : 0.0f;
    value = (value <= 255.0) ? value : 255.0f;
    return value;
}

/**
 * Color space transformation
 *
 * @param color_space_from
 * @param color_space_to
 */
template<enum gpujpeg_color_space color_space_from, enum gpujpeg_color_space color_space_to>
struct gpujpeg_color_transform
{
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        assert(false);
    }
};

/** Specialization [color_space_from = color_space_to] */
template<enum gpujpeg_color_space color_space>
struct gpujpeg_color_transform<color_space, color_space> {
    /** None transform */
    static __device__ void 
    perform(float & c1, float & c2, float & c3) {
        // Same color space so do nothing 
    }
};

/** Specialization [color_space_from = GPUJPEG_RGB, color_space_to = GPUJPEG_YCBCR_BT601] */
template<>
struct gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT601> {
    /** RGB -> YCbCr (ITU-R Recommendation BT.601) transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.equasys.de/colorconversion.html
        float r1 =  0.257000f * c1 + 0.504000f * c2 + 0.098000f * c3 + 16.0f;
        float r2 = -0.148000f * c1 - 0.291000f * c2 + 0.439000f * c3 + 128.0f;
        float r3 =  0.439000f * c1 - 0.368000f * c2 - 0.071000f * c3 + 128.0f;
        c1 = r1;
        c2 = r2;
        c3 = r3;
    }
};
/** Specialization [color_space_from = GPUJPEG_YCBCR_BT601, color_space_to = GPUJPEG_RGB] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT601, GPUJPEG_RGB> {
    /** YCbCr (ITU-R Recommendation BT.601) -> RGB transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.equasys.de/colorconversion.html
        float r1 = c1 - 16.0f;
        float r2 = c2 - 128.0f;
        float r3 = c3 - 128.0f;
        c1 = gpujpeg_clamp(1.164000f * r1 + 0.000000f * r2 + 1.596000f * r3);
        c2 = gpujpeg_clamp(1.164000f * r1 - 0.392000f * r2 - 0.813000f * r3);
        c3 = gpujpeg_clamp(1.164000f * r1 + 2.017000f * r2 + 0.000000f * r3);
    }
};

/** Specialization [color_space_from = GPUJPEG_RGB, color_space_to = GPUJPEG_YCBCR_BT601_256LVLS] */
template<>
struct gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT601_256LVLS> {
    /** RGB -> YCbCr (ITU-R Recommendation BT.601 with 256 levels) transform (8 bit) */
    static __device__ void 
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.ecma-international.org/publications/files/ECMA-TR/TR-098.pdf, page 3
        float r1 =  0.299000f * c1 + 0.587000f * c2 + 0.114000f * c3;
        float r2 = -0.168736f * c1 - 0.331264f * c2 + 0.500000f * c3 + 128.0f;
        float r3 =  0.500000f * c1 - 0.418688f * c2 - 0.081312f * c3 + 128.0f;
        c1 = r1;
        c2 = r2;
        c3 = r3;
    }
};
/** Specialization [color_space_from = GPUJPEG_YCBCR_BT601_256LVLS, color_space_to = GPUJPEG_RGB] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, GPUJPEG_RGB> {
    /** YCbCr (ITU-R Recommendation BT.601 with 256 levels) -> RGB transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.ecma-international.org/publications/files/ECMA-TR/TR-098.pdf, page 4
        float r1 = c1 - 0.0f;
        float r2 = c2 - 128.0f;
        float r3 = c3 - 128.0f;
        c1 = gpujpeg_clamp(1.000000f * r1 + 0.000000f * r2 + 1.402000f * r3);
        c2 = gpujpeg_clamp(1.000000f * r1 - 0.344136f * r2 - 0.714136f * r3);
        c3 = gpujpeg_clamp(1.000000f * r1 + 1.772000f * r2 + 0.000000f * r3);
    }
};

/** Specialization [color_space_from = GPUJPEG_RGB, color_space_to = GPUJPEG_YCBCR_BT709] */
template<>
struct gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT709> {
    /** RGB -> YCbCr (ITU-R Recommendation BT.709) transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.equasys.de/colorconversion.html
        float r1 =  0.183000f * c1 + 0.614000f * c2 + 0.062000f * c3 + 16.0f;
        float r2 = -0.101000f * c1 - 0.339000f * c2 + 0.439000f * c3 + 128.0f;
        float r3 =  0.439000f * c1 - 0.399000f * c2 - 0.040000f * c3 + 128.0f;
        c1 = r1;
        c2 = r2;
        c3 = r3;
    }
};
/** Specialization [color_space_from = GPUJPEG_YCBCR_BT709, color_space_to = GPUJPEG_RGB] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT709, GPUJPEG_RGB> {
    /** YCbCr (ITU-R Recommendation BT.709) -> RGB transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        // Source: http://www.equasys.de/colorconversion.html
        float r1 = c1 - 16.0f;
        float r2 = c2 - 128.0f;
        float r3 = c3 - 128.0f;
        c1 = gpujpeg_clamp(1.164000f * r1 + 0.000000f * r2 + 1.793000f * r3);
        c2 = gpujpeg_clamp(1.164000f * r1 - 0.213000f * r2 - 0.533000f * r3);
        c3 = gpujpeg_clamp(1.164000f * r1 + 2.112000f * r2 + 0.000000f * r3);
    }
};

/** Specialization [color_space_from = GPUJPEG_YCBCR_BT601, color_space_to = GPUJPEG_YCBCR_BT601_256LVLS] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT601, GPUJPEG_YCBCR_BT601_256LVLS> {
    /** YCbCr (ITU-R Recommendation BT.709) -> YCbCr (ITU-R Recommendation BT.601 with 256 levels) transform (8 bit) */
    static __device__ void 
    perform(float & c1, float & c2, float & c3) {
        gpujpeg_color_transform<GPUJPEG_YCBCR_BT601, GPUJPEG_RGB>::perform(c1,c2,c3);
        gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT601_256LVLS>::perform(c1,c2,c3);

    }
};
/** Specialization [color_space_from = GPUJPEG_YCBCR_BT601_256LVLS, color_space_to = GPUJPEG_YCBCR_BT601] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, GPUJPEG_YCBCR_BT601> {
    /** YCbCr (ITU-R Recommendation BT.601 with 256 levels) -> YCbCr (ITU-R Recommendation BT.709) transform (8 bit) */
    static __device__ void 
    perform(float & c1, float & c2, float & c3) {
        gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, GPUJPEG_RGB>::perform(c1,c2,c3);
        gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT601>::perform(c1,c2,c3);
    }
};

/** Specialization [color_space_from = GPUJPEG_YCBCR_BT709, color_space_to = GPUJPEG_YCBCR_BT601_256LVLS] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT709, GPUJPEG_YCBCR_BT601_256LVLS> {
    /** YCbCr (ITU-R Recommendation BT.709) -> YCbCr (ITU-R Recommendation BT.601 with 256 levels) transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        gpujpeg_color_transform<GPUJPEG_YCBCR_BT709, GPUJPEG_RGB>::perform(c1,c2,c3);
        gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT601_256LVLS>::perform(c1,c2,c3);

    }
};
/** Specialization [color_space_from = GPUJPEG_YCBCR_BT601_256LVLS, color_space_to = GPUJPEG_YCBCR_ITU_R] */
template<>
struct gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, GPUJPEG_YCBCR_BT709> {
    /** YCbCr (ITU-R Recommendation BT.601 with 256 levels) -> YCbCr (ITU-R Recommendation BT.709) transform (8 bit) */
    static __device__ void
    perform(float & c1, float & c2, float & c3) {
        gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, GPUJPEG_RGB>::perform(c1,c2,c3);
        gpujpeg_color_transform<GPUJPEG_RGB, GPUJPEG_YCBCR_BT709>::perform(c1,c2,c3);
    }
};

#define RGB_8BIT_THREADS 256

/**
 * Preprocessor data for component
 */
struct gpujpeg_preprocessor_data_component
{
    uint8_t* d_data;
    int data_width;
    struct gpujpeg_component_sampling_factor sampling_factor;
};

/**
 * Preprocessor data
 */
struct gpujpeg_preprocessor_data
{
    struct gpujpeg_preprocessor_data_component comp[3];
};

/** Value that means that sampling factor has dynamic value */
#define GPUJPEG_DYNAMIC 16

/** Sampling factor for all components */
typedef int gpujpeg_preprocessor_sampling_factor_t;

/**
 * Compose sampling factor for all components to single type
 * 
 * @return integer that contains all sampling factors
 */
inline gpujpeg_preprocessor_sampling_factor_t
gpujpeg_preprocessor_make_sampling_factor(int comp1_h, int comp1_v, int comp2_h, int comp2_v, int comp3_h, int comp3_v)
{    
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = 0;
    sampling_factor |= ((comp1_h << 4) | comp1_v) << 16;
    sampling_factor |= ((comp2_h << 4) | comp2_v) << 8;
    sampling_factor |= ((comp3_h << 4) | comp3_v) << 0;
    
    return sampling_factor;
}

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 */
template<
    uint8_t s_samp_factor_h = GPUJPEG_DYNAMIC,
    uint8_t s_samp_factor_v = GPUJPEG_DYNAMIC
>
struct gpujpeg_preprocessor_raw_to_comp_store
{
    static __device__ void
    perform(uint8_t value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        uint8_t samp_factor_h = s_samp_factor_h;
        if ( samp_factor_h == GPUJPEG_DYNAMIC ) {
            samp_factor_h = comp.sampling_factor.horizontal;
        }
        uint8_t samp_factor_v = s_samp_factor_v;
        if ( samp_factor_v == GPUJPEG_DYNAMIC ) {
            samp_factor_v = comp.sampling_factor.vertical;
        }
        
        if ( (position_x % samp_factor_h) != 0 && (position_x % samp_factor_v) != 0 )
            return;
        position_x = position_x / samp_factor_h;
        position_y = position_y / samp_factor_v;

        int data_position = position_y * comp.data_width + position_x;
        comp.d_data[data_position] = value;
    }
};
template<>
struct gpujpeg_preprocessor_raw_to_comp_store<1, 1>
{
    static __device__ void
    perform(uint8_t value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        int data_position = position_y * comp.data_width + position_x;
        comp.d_data[data_position] = value;
    }
};

/**
 * Kernel - Copy raw image source data into three separated component buffers
 */
typedef void (*gpujpeg_preprocessor_encode_kernel)(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, int image_width, int image_height);
 
/** Specialization [sampling factor is 4:4:4] */
template<
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void 
gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
            
    // Load to shared
    __shared__ unsigned char s_data[RGB_8BIT_THREADS * 3];
    if ( (x * 4) < RGB_8BIT_THREADS * 3 ) {
        int* s = (int*)d_data_raw;
        int* d = (int*)s_data;
        d[x] = s[((gX * 3) >> 2) + x];
    }
    __syncthreads();

    // Load
    int offset = x * 3;
    float r1 = (float)(s_data[offset]);
    float r2 = (float)(s_data[offset + 1]);
    float r3 = (float)(s_data[offset + 2]);

    // Color transform
    gpujpeg_color_transform<color_space, GPUJPEG_YCBCR_BT601_256LVLS>::perform(r1, r2, r3);
    
    // Position
    int image_position = gX + x;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
        
    // Store
    if ( image_position < (image_width * image_height) ) {
        gpujpeg_preprocessor_raw_to_comp_store<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform((uint8_t)r1, image_position_x, image_position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform((uint8_t)r2, image_position_x, image_position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform((uint8_t)r3, image_position_x, image_position_y, data.comp[2]);
    }
}

/** Specialization [sampling factor is 4:2:2] */
template<
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void 
gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
        
    // Load to shared
    __shared__ unsigned char s_data[RGB_8BIT_THREADS * 2];
    if ( (x * 4) < RGB_8BIT_THREADS * 2 ) {
        int* s = (int*)d_data_raw;
        int* d = (int*)s_data;
        d[x] = s[((gX * 2) >> 2) + x];
    }
    __syncthreads();

    // Load
    int offset = x * 2;
    float r1 = (float)(s_data[offset + 1]);
    float r2;
    float r3;
    if ( (gX + x) % 2 == 0 ) {
        r2 = (float)(s_data[offset]);
        r3 = (float)(s_data[offset + 2]);
    } else {
        r2 = (float)(s_data[offset - 2]);
        r3 = (float)(s_data[offset]);
    }
    
    // Color transform
    gpujpeg_color_transform<color_space, GPUJPEG_YCBCR_BT601_256LVLS>::perform(r1, r2, r3);
    
    // Position
    int image_position = gX + x;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
    
    // Store
    if ( image_position < (image_width * image_height) ) {
        gpujpeg_preprocessor_raw_to_comp_store<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform((uint8_t)r1, image_position_x, image_position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform((uint8_t)r2, image_position_x, image_position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform((uint8_t)r3, image_position_x, image_position_y, data.comp[2]);
    }
}

/**
 * Select preprocessor encode kernel
 * 
 * @param encoder
 * @return kernel
 */
gpujpeg_preprocessor_encode_kernel
gpujpeg_preprocessor_select_encode_kernel(struct gpujpeg_encoder* encoder)
{
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );
    
#define RETURN_KERNEL_IF(KERNEL, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose ) { \
            printf("Using faster kernel for preprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        return &KERNEL<COLOR, P1, P2, P3, P4, P5, P6>; \
    } 
#define RETURN_KERNEL(KERNEL, COLOR) \
    RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 1, 1, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 2, 2, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 2, 1, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 1, 2, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 4, 4, 4, 4) \
    else { \
        if ( coder->param.verbose ) { \
            printf("Using slower kernel for preprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        return &KERNEL<COLOR, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
    } \

    // RGB color space
    if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4);
        RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_RGB);
    } 
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT601);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT601);
        } else {
            assert(false);
        }
    } 
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT601_256LVLS);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT601_256LVLS);
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT709);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT709);
        } else {
            assert(false);
        }
    }
    // Unknown color space
    else {
        assert(false);
    }
    
#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL

    return NULL;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_encoder_init(struct gpujpeg_encoder* encoder)
{
    encoder->coder.preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel(encoder);
    if ( encoder->coder.preprocessor == NULL )
        return -1;
    return 0;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_encode(struct gpujpeg_encoder* encoder)
{    
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    hipMemset(coder->d_data, 0, coder->data_size * sizeof(uint8_t));

    // Select kernel
    gpujpeg_preprocessor_encode_kernel kernel = (gpujpeg_preprocessor_encode_kernel)coder->preprocessor;
    assert(kernel != NULL);
    
    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;
    
    // When loading 4:2:2 data of odd width, the data in fact has even width, so round it
    // (at least imagemagick convert tool generates data stream in this way)
    if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 )
        image_width = gpujpeg_div_and_round_up(coder->param_image.width, 2) * 2;
        
    // Prepare unit size
    assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4 || coder->param_image.sampling_factor == GPUJPEG_4_2_2);
    int unitSize = coder->param_image.sampling_factor == GPUJPEG_4_4_4 ? 3 : 2;
    
    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    if ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y = gpujpeg_div_and_round_up(grid.x, GPUJPEG_CUDA_MAXIMUM_GRID_SIZE);
        grid.x = GPUJPEG_CUDA_MAXIMUM_GRID_SIZE;
    }

    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < 3; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads>>>(
        data,
        coder->d_data_raw,
        image_width,
        image_height
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Preprocessor encoding failed");
        
    return 0;
}

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 * 
 * @param value
 * @param position_x
 * @param position_y
 * @param comp
 */
 template<
    uint8_t s_samp_factor_h = GPUJPEG_DYNAMIC,
    uint8_t s_samp_factor_v = GPUJPEG_DYNAMIC
>
struct gpujpeg_preprocessor_comp_to_raw_load
{
    static __device__ void
    perform(float & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        uint8_t samp_factor_h = s_samp_factor_h;
        if ( samp_factor_h == GPUJPEG_DYNAMIC ) {
            samp_factor_h = comp.sampling_factor.horizontal;
        }
        uint8_t samp_factor_v = s_samp_factor_v;
        if ( samp_factor_v == GPUJPEG_DYNAMIC ) {
            samp_factor_v = comp.sampling_factor.vertical;
        }
        
        position_x = position_x / samp_factor_h;
        position_y = position_y / samp_factor_v;
        
        int data_position = position_y * comp.data_width + position_x;
        value = (float)comp.d_data[data_position];
    }
};
template<>
struct gpujpeg_preprocessor_comp_to_raw_load<1, 1>
{
    static __device__ void
    perform(float & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        int data_position = position_y * comp.data_width + position_x;
        value = (float)comp.d_data[data_position];
    }
};

/**
 * Kernel - Copy three separated component buffers into target image data
 *
 * @param d_c1  First component buffer
 * @param d_c2  Second component buffer
 * @param d_c3  Third component buffer
 * @param d_target  Image target data
 * @param pixel_count  Number of pixels to copy
 * @return void
 */
typedef void (*gpujpeg_preprocessor_decode_kernel)(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height);

/** Specialization [sampling factor is 4:4:4] */
template<
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int image_position = gX + x;
    if ( image_position >= (image_width * image_height) )
        return;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
        
    // Load
    float r1;
    float r2;
    float r3;
    gpujpeg_preprocessor_comp_to_raw_load<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform(r1, image_position_x, image_position_y, data.comp[0]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform(r2, image_position_x, image_position_y, data.comp[1]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r3, image_position_x, image_position_y, data.comp[2]);
    
    // Color transform
    gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, color_space>::perform(r1, r2, r3);
    
    // Save
    image_position = image_position * 3;
    d_data_raw[image_position + 0] = (uint8_t)round(r1);
    d_data_raw[image_position + 1] = (uint8_t)round(r2);
    d_data_raw[image_position + 2] = (uint8_t)round(r3);
}

/** Specialization [sampling factor is 4:2:2] */
template<
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int image_position = gX + x;
    if ( image_position >= (image_width * image_height) )
        return;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
        
    // Load
    float r1;
    float r2;
    float r3;
    gpujpeg_preprocessor_comp_to_raw_load<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform(r1, image_position_x, image_position_y, data.comp[0]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform(r2, image_position_x, image_position_y, data.comp[1]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r3, image_position_x, image_position_y, data.comp[2]);
    
    // Color transform
    gpujpeg_color_transform<GPUJPEG_YCBCR_BT601_256LVLS, color_space>::perform(r1, r2, r3);
    
    // Save
    image_position = image_position * 2;
    d_data_raw[image_position + 1] = (uint8_t)round(r1);
    if ( (image_position_x % 2) == 0 )
        d_data_raw[image_position + 0] = (uint8_t)round(r2);
    else
        d_data_raw[image_position + 0] = (uint8_t)round(r3);
}

/**
 * Select preprocessor decode kernel
 * 
 * @param decoder
 * @return kernel
 */
gpujpeg_preprocessor_decode_kernel
gpujpeg_preprocessor_select_decode_kernel(struct gpujpeg_decoder* decoder)
{
    // Get coder
    struct gpujpeg_coder* coder = &decoder->coder;
    
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );
    
#define RETURN_KERNEL_IF(KERNEL, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose ) { \
            printf("Using faster kernel for postprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        return &KERNEL<COLOR, P1, P2, P3, P4, P5, P6>; \
    } 
#define RETURN_KERNEL(KERNEL, COLOR) \
    RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 1, 1, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 2, 2, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 2, 1, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 1, 2, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 4, 4, 4, 4) \
    else { \
        if ( coder->param.verbose ) { \
            printf("Using slower kernel for postprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        return &KERNEL<COLOR, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
    } \
    
    // RGB color space
    if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4);
        RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_RGB);
    } 
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT601)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT601)
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT601_256LVLS)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT601_256LVLS)
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT709)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT709)
        } else {
            assert(false);
        }
    }
    // Unknown color space
    else {
        assert(false);
    }
    
#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL
    
    return NULL;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_decoder_init(struct gpujpeg_decoder* decoder)
{
    decoder->coder.preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel(decoder);
    if ( decoder->coder.preprocessor == NULL )
        return -1;
    return 0;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_decode(struct gpujpeg_decoder* decoder)
{
    // Get coder
    struct gpujpeg_coder* coder = &decoder->coder;
    
    hipMemset(coder->d_data_raw, 0, coder->data_raw_size * sizeof(uint8_t));
    
    // Select kernel
    gpujpeg_preprocessor_decode_kernel kernel = (gpujpeg_preprocessor_decode_kernel)coder->preprocessor;
    assert(kernel != NULL);
    
    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;
    
    // When saving 4:2:2 data of odd width, the data should have even width, so round it
    if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 )
        image_width = gpujpeg_div_and_round_up(coder->param_image.width, 2) * 2;
        
    // Prepare unit size
    assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4 || coder->param_image.sampling_factor == GPUJPEG_4_2_2);
    int unitSize = coder->param_image.sampling_factor == GPUJPEG_4_4_4 ? 3 : 2;
    
    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    if ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y = gpujpeg_div_and_round_up(grid.x, GPUJPEG_CUDA_MAXIMUM_GRID_SIZE);
        grid.x = GPUJPEG_CUDA_MAXIMUM_GRID_SIZE;
    }

    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < 3; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads>>>(
        data,
        coder->d_data_raw, 
        image_width,
        image_height
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Preprocessor encoding failed");
    
    return 0;
}
