#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "gpujpeg_dct_gpu.h"
#include "gpujpeg_util.h"

/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/** Fast integer multiplication */
#define FMUL(x,y)   (__mul24(x,y))
//#define FMUL(x,y)   ((x)*(y))

// X block count which will be processed by one thread block
#define GPUJPEG_DCT_BLOCK_COUNT_X       4
// Y block count which will be processed by one thread block
#define GPUJPEG_DCT_BLOCK_COUNT_Y       4

// Thread block width
#define GPUJPEG_DCT_THREAD_BLOCK_WIDTH  (GPUJPEG_BLOCK_SIZE * GPUJPEG_DCT_BLOCK_COUNT_X)
// Thread block height
#define GPUJPEG_DCT_THREAD_BLOCK_HEIGHT (GPUJPEG_BLOCK_SIZE * GPUJPEG_DCT_BLOCK_COUNT_Y)

// Stride of shared memory buffer (short kernel)
#define GPUJPEG_DCT_THREAD_BLOCK_STRIDE (GPUJPEG_DCT_THREAD_BLOCK_WIDTH + 4)

#define IMAD(a, b, c) ( ((a) * (b)) + (c) )
#define IMUL(a, b) ((a) * (b))

#define SIN_1_4     0x5A82
#define COS_1_4     0x5A82
#define SIN_1_8     0x30FC
#define COS_1_8     0x7642

#define OSIN_1_16   0x063E
#define OSIN_3_16   0x11C7
#define OSIN_5_16   0x1A9B
#define OSIN_7_16   0x1F63

#define OCOS_1_16   0x1F63
#define OCOS_3_16   0x1A9B
#define OCOS_5_16   0x11C7
#define OCOS_7_16   0x063E

/**
 * Package of 2 shorts into 1 int - designed to perform i/o by integers to avoid bank conflicts
 */
union PackedInteger
{
    struct __align__(8)
    {
        int16_t hShort1;
        int16_t hShort2;
    };
    int32_t hInt;
};

/**
 * Converts fixed point value to short value
 */
__device__ inline int16_t
unfixh(int x)
{
    return (int16_t)((x + 0x8000) >> 16);
}

/**
 * Converts fixed point value to short value
 */
__device__ inline int
unfixo(int x)
{
    return (x + 0x1000) >> 13;
}


template <typename T>
__device__ static inline void
dct(const T in0, const T in1, const T in2, const T in3, const T in4, const T in5, const T in6, const T in7,
    volatile T & out0, volatile T & out1, volatile T & out2, volatile T & out3, volatile T & out4, volatile T & out5, volatile T & out6, volatile T & out7,
    const float level_shift = 0.0f)
{
//     const int tmp0 = in7 + in0;
//     const int tmp1 = in6 + in1;
//     const int tmp2 = in5 + in2;
//     const int tmp3 = in4 + in3;
//     const int tmp4 = in3 - in4;
//     const int tmp5 = in2 - in5;
//     const int tmp6 = in1 - in6;
//     const int tmp7 = in0 - in7;
// 
//     const int tmp10 = tmp3 + tmp0;
//     const int tmp11 = tmp2 + tmp1;
//     const int tmp12 = tmp1 - tmp2;
//     const int tmp13 = tmp0 - tmp3;
// 
//     const int tmp16 = unfixo(FMUL(tmp6 + tmp5, SIN_1_4));
//     const int tmp15 = unfixo(FMUL(tmp6 - tmp5, COS_1_4));
// 
//     const int tmp4b = tmp4 << 2;
//     const int tmp7b = tmp7 << 2;
// 
//     const int tmp14 = tmp4b + tmp15;
//     const int tmp25 = tmp4b - tmp15;
//     const int tmp26 = tmp7b - tmp16;
//     const int tmp17 = tmp7b + tmp16;
//     
//     out0 = unfixh(FMUL(tmp10 + tmp11, SIN_1_4));
//     out1 = unfixh(FMUL(tmp17, OCOS_1_16) + FMUL(tmp14, OSIN_1_16));
//     out2 = unfixh(FMUL(tmp13, COS_1_8) + FMUL(tmp12, SIN_1_8));
//     out3 = unfixh(FMUL(tmp26, OCOS_3_16) - FMUL(tmp25, OSIN_3_16));
//     out4 = unfixh(FMUL(tmp10 - tmp11, COS_1_4));
//     out5 = unfixh(FMUL(tmp26, OCOS_5_16) + FMUL(tmp25, OSIN_5_16));
//     out6 = unfixh(FMUL(tmp13, SIN_1_8) - FMUL(tmp12, COS_1_8));
//     out7 = unfixh(FMUL(tmp17, OCOS_7_16) - FMUL(tmp14, OSIN_7_16));


//     const float scale0 = 0.353553390593274f; // sin(pi / 4) / 2
//     const float scale1 = 0.509795579104159f; // 1 / (2 * sin(7 * pi / 16))
//     const float scale2 = 0.541196100146197f; // 1 / (2 * sin(3 * pi / 8))
//     const float scale3 = 0.601344886935045f; // 1 / (2 * cos(3 * pi / 16))
//     const float scale4 = 0.707106781186547f; // sin(pi / 4)
//     const float scale5 = 0.415734806151273f; // cos(3 * pi / 16) / 2
//     const float scale6 = 0.461939766255643f; // sin(3 * pi / 8) / 2
//     const float scale7 = 0.490392640201615f; // sin(7 * pi / 16) / 2
//     
//     const float p1 = 0.4142135623f;
//     const float p2 = 0.6681786379f;
//     const float p3 = 0.1989123673f;
//     const float p4 = 0.4142135623f;
//     const float p5 = 0.4142135623f;
//     const float u1 = 0.3535533905f;
//     const float u2 = 0.4619397662f;
//     const float u3 = 0.1913417161f;
//     const float u4 = 0.7071067811f;
//     
//     float a0 = in7 + in0;
//     float a1 = in6 + in1;
//     float a2 = in5 + in2;
//     float a3 = in4 + in3;
//     float a4 = in3 - in4;
//     float a5 = in2 - in5;
//     float a6 = in1 - in6;
//     float a7 = in0 - in7;
//     
//     a5 = a5 - a6 * p4;
//     a6 = a6 + a5 * u4;
//     a5 = a6 * p5 - a5;
//     
//     float b0 = a0 + a3;
//     float b1 = a1 + a2;
//     float b2 = a1 - a2;
//     float b3 = a0 - a3;
//     float b4 = a4 + a5;
//     float b5 = a4 - a5;
//     float b6 = a7 - a6;
//     float b7 = a7 + a6;
//     
//     b0 = b0 + b1;
//     b1 = 0.5f * b0 - b1;
//     
//     b2 = p1 * b3 - b2;
//     b3 = b3 - u1 * b2;
//     
//     b4 = p3 * b7 - b4;
//     b7 = b7 - u3 * b4;
//     
//     b5 = b5 + p2 * b6;
//     b6 = b6 - u2 * b5;
//     
//     out0 = b0 * scale0;
//     out1 = b7 * scale7;
//     out2 = b3 * scale3;
//     out3 = b6 * scale6;
//     out4 = b1 * scale1;
//     out5 = b5 * scale5;
//     out6 = b2 * scale2;
//     out7 = b4 * scale4;
    
    
    /* Load data into workspace */
    const float tmp0 = in0 + in7;
    const float tmp7 = in0 - in7;
    const float tmp1 = in1 + in6;
    const float tmp6 = in1 - in6;
    const float tmp2 = in2 + in5;
    const float tmp5 = in2 - in5;
    const float tmp3 = in3 + in4;
    const float tmp4 = in3 - in4;

    {
        /* Even part */

        const float tmp10 = tmp0 + tmp3;        /* phase 2 */
        const float tmp13 = tmp0 - tmp3;
        const float tmp11 = tmp1 + tmp2;
        const float tmp12 = tmp1 - tmp2;

        /* Apply unsigned->signed conversion */
        out0 = tmp10 + tmp11 - 8 * level_shift; /* phase 3 */
        out4 = tmp10 - tmp11;

        const float z1 = (tmp12 + tmp13) * 0.707106781f; /* c4 */
        out2 = tmp13 + z1;    /* phase 5 */
        out6 = tmp13 - z1;
    }

    
    /* Odd part */

    const float tmp10 = tmp4 + tmp5;        /* phase 2 */
    const float tmp11 = tmp5 + tmp6;
    const float tmp12 = tmp6 + tmp7;

    /* The rotator is modified from fig 4-8 to avoid extra negations. */
    const float z5 = (tmp10 - tmp12) * 0.382683433f; /* c6 */
    const float z2 = 0.541196100f * tmp10 + z5; /* c2-c6 */
    const float z4 = 1.306562965f * tmp12 + z5; /* c2+c6 */
    const float z3 = tmp11 * 0.707106781f; /* c4 */

    const float z11 = tmp7 + z3;            /* phase 5 */
    const float z13 = tmp7 - z3;

    out5 = z13 + z2;      /* phase 6 */
    out3 = z13 - z2;
    out1 = z11 + z4;
    out7 = z11 - z4;
}


/**
 * Performs in-place DCT of vector of 8 elements (used to access columns in shared memory).
 *
 * @param SrcDst [IN/OUT] - Pointer to the first element of vector
 * @param Stride [IN] - Value to add to ptr to access other elements
 * @return None
 */
__device__ void
gpujpeg_dct_gpu_kernel_inplace(float* SrcDst, int Stride)
{
    dct(SrcDst[Stride * 0], SrcDst[Stride * 1], SrcDst[Stride * 2], SrcDst[Stride * 3],
        SrcDst[Stride * 4], SrcDst[Stride * 5], SrcDst[Stride * 6], SrcDst[Stride * 7],
        SrcDst[Stride * 0], SrcDst[Stride * 1], SrcDst[Stride * 2], SrcDst[Stride * 3],
        SrcDst[Stride * 4], SrcDst[Stride * 5], SrcDst[Stride * 6], SrcDst[Stride * 7]);
}


/**
 * Performs in-place IDCT of vector of 8 elements (used to access columns in shared memory).
 *
 * @param SrcDst [IN/OUT] - Pointer to the first element of vector
 * @param Stride [IN] - Value to add to ptr to access other elements
 * @return None
 */
__device__ void
gpujpeg_idct_gpu_kernel_inplace(int16_t* SrcDst, int Stride)
{
    int in0, in1, in2, in3, in4, in5, in6, in7;
    int tmp10, tmp11, tmp12, tmp13;
    int tmp20, tmp21, tmp22, tmp23;
    int tmp30, tmp31;
    int tmp40, tmp41, tmp42, tmp43;
    int tmp50, tmp51, tmp52, tmp53;

    int16_t *DstPtr = SrcDst;
    in0 = *DstPtr;
    DstPtr += Stride;
    in1 = *DstPtr;
    DstPtr += Stride;
    in2 = *DstPtr;
    DstPtr += Stride;
    in3 = *DstPtr;
    DstPtr += Stride;
    in4 = *DstPtr;
    DstPtr += Stride;
    in5 = *DstPtr;
    DstPtr += Stride;
    in6 = *DstPtr;
    DstPtr += Stride;
    in7 = *DstPtr;

    tmp10 = FMUL(in0 + in4, COS_1_4);
    tmp11 = FMUL(in0 - in4, COS_1_4);
    tmp12 = FMUL(in2, SIN_1_8) - FMUL(in6, COS_1_8);
    tmp13 = FMUL(in6, SIN_1_8) + FMUL(in2, COS_1_8);

    tmp20 = tmp10 + tmp13;
    tmp21 = tmp11 + tmp12;
    tmp22 = tmp11 - tmp12;
    tmp23 = tmp10 - tmp13;

    tmp30 = unfixo(FMUL(in3 + in5, COS_1_4));
    tmp31 = unfixo(FMUL(in3 - in5, COS_1_4));

    in1 <<= 2;
    in7 <<= 2;

    tmp40 = in1 + tmp30;
    tmp41 = in7 + tmp31;
    tmp42 = in1 - tmp30;
    tmp43 = in7 - tmp31;

    tmp50 = FMUL(tmp40, OCOS_1_16) + FMUL(tmp41, OSIN_1_16);
    tmp51 = FMUL(tmp40, OSIN_1_16) - FMUL(tmp41, OCOS_1_16);
    tmp52 = FMUL(tmp42, OCOS_5_16) + FMUL(tmp43, OSIN_5_16);
    tmp53 = FMUL(tmp42, OSIN_5_16) - FMUL(tmp43, OCOS_5_16);

    DstPtr = SrcDst;
    *DstPtr = unfixh(tmp20 + tmp50);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp21 + tmp53);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp22 + tmp52);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp23 + tmp51);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp23 - tmp51);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp22 - tmp52);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp21 - tmp53);
    DstPtr += Stride;
    *DstPtr = unfixh(tmp20 - tmp50);
}

/**
 * Performs in-place IDCT of vector of 8 elements (used to access rows in shared memory).
 *
 * @param V8 [IN/OUT] - Pointer to the first two elements of vector
 * @return None
 */
__device__ void
gpujpeg_idct_gpu_kernel_inplace(uint32_t* V8)
{
    int in0, in1, in2, in3, in4, in5, in6, in7;
    int tmp10, tmp11, tmp12, tmp13;
    int tmp20, tmp21, tmp22, tmp23;
    int tmp30, tmp31;
    int tmp40, tmp41, tmp42, tmp43;
    int tmp50, tmp51, tmp52, tmp53;
    PackedInteger sh0, sh1, sh2, sh3;

    sh0.hInt = V8[0];
    sh1.hInt = V8[1];
    sh2.hInt = V8[2];
    sh3.hInt = V8[3];
    in0 = sh0.hShort1;
    in1 = sh0.hShort2;
    in2 = sh1.hShort1;
    in3 = sh1.hShort2;
    in4 = sh2.hShort1;
    in5 = sh2.hShort2;
    in6 = sh3.hShort1;
    in7 = sh3.hShort2;

    tmp10 = FMUL(in0 + in4, COS_1_4);
    tmp11 = FMUL(in0 - in4, COS_1_4);
    tmp12 = FMUL(in2, SIN_1_8) - FMUL(in6, COS_1_8);
    tmp13 = FMUL(in6, SIN_1_8) + FMUL(in2, COS_1_8);

    tmp20 = tmp10 + tmp13;
    tmp21 = tmp11 + tmp12;
    tmp22 = tmp11 - tmp12;
    tmp23 = tmp10 - tmp13;

    tmp30 = unfixo(FMUL(in3 + in5, COS_1_4));
    tmp31 = unfixo(FMUL(in3 - in5, COS_1_4));

    in1 <<= 2;
    in7 <<= 2;

    tmp40 = in1 + tmp30;
    tmp41 = in7 + tmp31;
    tmp42 = in1 - tmp30;
    tmp43 = in7 - tmp31;

    tmp50 = FMUL(tmp40, OCOS_1_16) + FMUL(tmp41, OSIN_1_16);
    tmp51 = FMUL(tmp40, OSIN_1_16) - FMUL(tmp41, OCOS_1_16);
    tmp52 = FMUL(tmp42, OCOS_5_16) + FMUL(tmp43, OSIN_5_16);
    tmp53 = FMUL(tmp42, OSIN_5_16) - FMUL(tmp43, OCOS_5_16);

    sh0.hShort1 = unfixh(tmp20 + tmp50);
    sh0.hShort2 = unfixh(tmp21 + tmp53);
    sh1.hShort1 = unfixh(tmp22 + tmp52);
    sh1.hShort2 = unfixh(tmp23 + tmp51);
    sh2.hShort1 = unfixh(tmp23 - tmp51);
    sh2.hShort2 = unfixh(tmp22 - tmp52);
    sh3.hShort1 = unfixh(tmp21 - tmp53);
    sh3.hShort2 = unfixh(tmp20 - tmp50);

    V8[0] = sh0.hInt;
    V8[1] = sh1.hInt;
    V8[2] = sh2.hInt;
    V8[3] = sh3.hInt;
}

/** Quantization table */
#if __CUDA_ARCH__ < 200
__constant__ // quantization table in constant mempory is faster on devices without L2 cache
#endif
__device__ float gpujpeg_dct_gpu_quantization_table[64];

/**
 * Performs 8x8 block-wise Forward Discrete Cosine Transform of the given
 * image plane and outputs result to the array of coefficients. Short implementation.
 * This kernel is designed to process image by blocks of blocks8x8 that
 * utilize maximum warps capacity, assuming that it is enough of 8 threads
 * per block8x8.
 *
 * @param source        [IN]  - Source coefficients
 * @param source_stride [IN]  - Stride of source
 * @param output        [OUT] - Source coefficients
 * @param output_stride [OUT] - Stride of source
 * @return None
 */
template <int WARP_COUNT>
__global__ void
gpujpeg_dct_gpu_kernel(int block_count_x, int block_count_y, uint8_t* source, int source_stride,
                       int16_t* output, int output_stride)
{
    // each warp processes 4 8x8 blocks (horizontally neighboring)
    const int block_idx_x = threadIdx.x >> 3;
    const int block_idx_y = threadIdx.y;
    
    // offset of threadblocks's blocks in the image (along both axes)
    const int block_offset_x = blockIdx.x * 4;
    const int block_offset_y = blockIdx.y * WARP_COUNT;
    
    // true if thread's block is not out of image
    const bool processing = block_offset_x + block_idx_x < block_count_x
                         && block_offset_y + block_idx_y < block_count_y;
    
    // stop if out of block range
    if(!processing) {
        return;
    }
    
    // index of row/column processed by this thread within its 8x8 block
    const int dct_idx = threadIdx.x & 7;
    
    
    
    // Data type of transformed coefficients
    typedef float dct_t;
    
    // dimensions of shared buffer (compile time constants)
    enum {
        // 4 8x8 blocks, padded to odd number of 4byte banks
        SHARED_STRIDE = ((32 * sizeof(dct_t)) | 4) / sizeof(dct_t),
        
        // number of shared buffer items needed for 1 warp
        SHARED_SIZE_WARP = SHARED_STRIDE * 8,
        
        // total number of items in shared buffer
        SHARED_SIZE_TOTAL = SHARED_SIZE_WARP * WARP_COUNT
    };
    
    // buffer for transpositions of all blocks
    __shared__ volatile dct_t s_transposition_all[SHARED_SIZE_TOTAL];
    
    // pointer to begin of transposition buffer for thread's block
    volatile dct_t * const s_transposition = s_transposition_all + block_idx_y * SHARED_SIZE_WARP + block_idx_x * 8;
    
    
    
    
    
    
    // Load input coefficients (each thread loads 1 row of 8 coefficients from its 8x8 block)
    const int in_x = (block_offset_x + block_idx_x) * 8 + dct_idx;
    const int in_y = (block_offset_y + block_idx_y) * 8;
    const int in_offset = in_x + in_y * source_stride;
    const uint8_t * in = source + in_offset;
    
    // separate input coefficients and apply level shift (assuming little endian hardware)
    dct_t src0 = *in;
    in += source_stride;
    dct_t src1 = *in;
    in += source_stride;
    dct_t src2 = *in;
    in += source_stride;
    dct_t src3 = *in;
    in += source_stride;
    dct_t src4 = *in;
    in += source_stride;
    dct_t src5 = *in;
    in += source_stride;
    dct_t src6 = *in;
    in += source_stride;
    dct_t src7 = *in;
    
    
    
    
    
        // destination pointer into shared transpose buffer (each thread saves one column)
    volatile dct_t * const s_dest = s_transposition + dct_idx;
    
    dct(src0, src1, src2, src3, src4, src5, src6, src7,
        s_dest[SHARED_STRIDE * 0],
        s_dest[SHARED_STRIDE * 1],
        s_dest[SHARED_STRIDE * 2],
        s_dest[SHARED_STRIDE * 3],
        s_dest[SHARED_STRIDE * 4],
        s_dest[SHARED_STRIDE * 5],
        s_dest[SHARED_STRIDE * 6],
        s_dest[SHARED_STRIDE * 7],
        128
    );
    
    // read coefficients back - each thread reads one row (no need to sync - only threads within same warp work on each block)
    volatile dct_t * s_src = s_transposition + SHARED_STRIDE * dct_idx;
    dct_t dct0, dct1, dct2, dct3, dct4, dct5, dct6, dct7;
    dct(s_src[0], s_src[1], s_src[2], s_src[3], s_src[4], s_src[5], s_src[6], s_src[7],
        dct0, dct1, dct2, dct3, dct4, dct5, dct6, dct7);
    
    
    // apply qunatzation to the row of coefficients
    const float * const quantization_row = gpujpeg_dct_gpu_quantization_table + 8 * dct_idx;
    const int out0 = 0.5f + dct0 * quantization_row[0];
    const int out1 = 0.5f + dct1 * quantization_row[1];
    const int out2 = 0.5f + dct2 * quantization_row[2];
    const int out3 = 0.5f + dct3 * quantization_row[3];
    const int out4 = 0.5f + dct4 * quantization_row[4];
    const int out5 = 0.5f + dct5 * quantization_row[5];
    const int out6 = 0.5f + dct6 * quantization_row[6];
    const int out7 = 0.5f + dct7 * quantization_row[7];
    
    // save output row packed into 16 bytes
    const int out_x = (block_offset_x + block_idx_x) * 64; // 64 coefficients per one transformed and quantized block
    const int out_y = (block_offset_y + block_idx_y) * output_stride;
    ((uint4*)(output + out_x + out_y))[dct_idx] = make_uint4(
        out0 + 0x10000 * out1,
        out2 + 0x10000 * out3,
        out4 + 0x10000 * out5,
        out6 + 0x10000 * out7
    );
    
    
    
    
//     // Shared data
//     __shared__ float block[GPUJPEG_DCT_THREAD_BLOCK_HEIGHT * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];
// 
//     // Block position
//     int block_x = IMAD(blockIdx.x, GPUJPEG_DCT_BLOCK_COUNT_X, threadIdx.y);
//     int block_y = IMAD(blockIdx.y, GPUJPEG_DCT_BLOCK_COUNT_Y, threadIdx.z);
// 
//     // Thread position in thread block
//     int thread_x = IMAD(threadIdx.y, GPUJPEG_BLOCK_SIZE, threadIdx.x);
//     int thread_y = IMUL(threadIdx.z, GPUJPEG_BLOCK_SIZE);
//     int thread_x_permutated = (thread_x & 0xFFFFFFE0) | (((thread_x << 1) | ((thread_x >> 4) & 0x1)) & 0x1F);
// 
//     // Determine position into shared buffer
//     float* block_ptr = block + IMAD(thread_y, GPUJPEG_DCT_THREAD_BLOCK_STRIDE, thread_x);
// 
//     // Determine position in source buffer and apply it
//     int source_x = IMAD(block_x, GPUJPEG_BLOCK_SIZE, threadIdx.x);
//     int source_y = IMUL(block_y, GPUJPEG_BLOCK_SIZE);
//     source += IMAD(source_y, source_stride, source_x);
// 
//     // Load data to shared memory memory
//     if ( block_x < block_count_x && block_y < block_count_y ) {
//         #pragma unroll
//         for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++) {
//             float coefficient = (int16_t)(source[i * source_stride]);
//             coefficient -= 128.0f;
//             block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE] = coefficient;
//         }
//     }
// 
//     // Perform DCT
//     __syncthreads();
//     gpujpeg_dct_gpu_kernel_inplace(block + thread_y * GPUJPEG_DCT_THREAD_BLOCK_STRIDE + thread_x_permutated, GPUJPEG_DCT_THREAD_BLOCK_STRIDE);
//     __syncthreads();
//     gpujpeg_dct_gpu_kernel_inplace(block + (thread_y + threadIdx.x) * GPUJPEG_DCT_THREAD_BLOCK_STRIDE + threadIdx.y * GPUJPEG_BLOCK_SIZE, 1);
//     __syncthreads();
// 
//     // Quantization
//     for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++) {
//         float quantization = (quantization_table[i * GPUJPEG_BLOCK_SIZE + threadIdx.x]) / 32767.0f;
//         float coefficient = block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];
//         block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE] = coefficient * quantization;
//     }
//     __syncthreads();
// 
//     // Determine position in output buffer and apply it
//     int output_x = IMAD(IMAD(blockIdx.x, GPUJPEG_DCT_BLOCK_COUNT_X, threadIdx.y), GPUJPEG_BLOCK_SQUARED_SIZE, threadIdx.x);
//     int output_y = IMAD(blockIdx.y, GPUJPEG_DCT_BLOCK_COUNT_Y, threadIdx.z);
//     output += IMAD(output_y, output_stride, output_x);
// 
//     // Store data to global memory
//     if ( block_x < block_count_x && block_y < block_count_y ) {
//         #pragma unroll
//         for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++)
//             output[i * GPUJPEG_BLOCK_SIZE] = round(block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE]);
//     }
}

/** Quantization table */
__constant__ uint16_t gpujpeg_idct_gpu_quantization_table[64];

/**
 * Performs 8x8 block-wise Inverse Discrete Cosine Transform of the given
 * image plane and outputs result to the array of coefficients. Short implementation.
 * This kernel is designed to process image by blocks of blocks8x8 that
 * utilize maximum warps capacity, assuming that it is enough of 8 threads
 * per block8x8.
 *
 * @param source        [IN]  - Source coefficients
 * @param source_stride [IN]  - Stride of source
 * @param output        [OUT] - Source coefficients
 * @param output_stride [OUT] - Stride of source
 * @param table         [IN]  - Quantization table
 * @return None
 */
__global__ void
gpujpeg_idct_gpu_kernel(int block_count_x, int block_count_y, int16_t* source, int source_stride,
                        uint8_t* output, int output_stride, uint16_t* quantization_table)
{
// For pre-fermi GPUs, quantization table in constant memory is faster
#if __CUDA_ARCH__ < 200
    quantization_table = gpujpeg_idct_gpu_quantization_table;
#endif
    
    // Shared data
    __shared__ int16_t block[GPUJPEG_DCT_THREAD_BLOCK_HEIGHT * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];

    // Block position
    int block_x = IMAD(blockIdx.x, GPUJPEG_DCT_BLOCK_COUNT_X, threadIdx.y);
    int block_y = IMAD(blockIdx.y, GPUJPEG_DCT_BLOCK_COUNT_Y, threadIdx.z);

    // Thread position in thread block
    int thread_x = IMAD(threadIdx.y, GPUJPEG_BLOCK_SIZE, threadIdx.x);
    int thread_y = IMUL(threadIdx.z, GPUJPEG_BLOCK_SIZE);
    int thread_x_permutated = (thread_x & 0xFFFFFFE0) | (((thread_x << 1) | ((thread_x >> 4) & 0x1)) & 0x1F);

    // Determine position into shared buffer
    int16_t* block_ptr = block + IMAD(thread_y, GPUJPEG_DCT_THREAD_BLOCK_STRIDE, thread_x);

    // Determine position in source buffer and apply it    
    int source_x = IMAD(block_x, GPUJPEG_BLOCK_SQUARED_SIZE, threadIdx.x * 2);
    int source_y = block_y;
    source += IMAD(source_y, source_stride, source_x);

    // Load data to shared memory, only half of threads in each cell performs data moving (each thread moves 2 shorts)
    if ( block_x < block_count_x && block_y < block_count_y ) {
        int16_t* block_load_ptr = block_ptr + threadIdx.x; // Shortcut for "IMAD(..., threadIdx.x * 2)"
        if ( threadIdx.x < (GPUJPEG_BLOCK_SIZE / 2) ) {
            #pragma unroll
            for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++)
                ((int*)block_load_ptr)[i * (GPUJPEG_DCT_THREAD_BLOCK_STRIDE / 2)] = ((int*)source)[i * (GPUJPEG_BLOCK_SIZE / 2)];
        }
    }
    __syncthreads();

    // Quantization
    for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++) {
        int16_t quantization = quantization_table[i * GPUJPEG_BLOCK_SIZE + threadIdx.x];
        int16_t coefficient = block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];

        coefficient = coefficient * quantization;

        block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE] = coefficient;
    }

    // Perform IDCT
    __syncthreads();
    gpujpeg_idct_gpu_kernel_inplace(block + thread_y * GPUJPEG_DCT_THREAD_BLOCK_STRIDE + thread_x_permutated, GPUJPEG_DCT_THREAD_BLOCK_STRIDE);
    __syncthreads();
    gpujpeg_idct_gpu_kernel_inplace((uint32_t*)(block + (thread_y + threadIdx.x) * GPUJPEG_DCT_THREAD_BLOCK_STRIDE + threadIdx.y * GPUJPEG_BLOCK_SIZE));
    __syncthreads();

     // Determine position in output buffer and apply it
    int output_x = IMAD(blockIdx.x, GPUJPEG_DCT_THREAD_BLOCK_WIDTH, thread_x);
    int output_y = IMAD(blockIdx.y, GPUJPEG_DCT_THREAD_BLOCK_HEIGHT, thread_y);
    output += IMAD(output_y, output_stride, output_x);

// For pre-fermi GPUs, storing to global memory by 4 bytes is faster
#if __CUDA_ARCH__ < 200
    __shared__ uint8_t block_byte[GPUJPEG_DCT_THREAD_BLOCK_HEIGHT * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];
    uint8_t* block_byte_ptr = block_byte + IMAD(thread_y, GPUJPEG_DCT_THREAD_BLOCK_STRIDE, thread_x);
    uint8_t* __output = output;
    int __output_stride = output_stride;
    output = block_byte_ptr;
    output_stride = GPUJPEG_DCT_THREAD_BLOCK_STRIDE;
#endif

    // Store data to global memory
    if ( block_x < block_count_x && block_y < block_count_y ) {
        #pragma unroll
        for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++) {
            int16_t coefficient = block_ptr[i * GPUJPEG_DCT_THREAD_BLOCK_STRIDE];
            coefficient += 128;
            if ( coefficient > 255 )
                coefficient = 255;
            if ( coefficient < 0 )
                coefficient = 0;
            output[i * output_stride] = (uint8_t)coefficient;
        }
        
// For pre-fermi GPUs, storing to global memory by 4 bytes is faster
#if __CUDA_ARCH__ < 200
        if ( threadIdx.x % 4 == 0 ) {
            #pragma unroll
            for(int i = 0; i < GPUJPEG_BLOCK_SIZE; i++)
                ((uint32_t*)__output)[i * (__output_stride / 4)] = ((uint32_t*)block_byte_ptr)[i * (GPUJPEG_DCT_THREAD_BLOCK_STRIDE / 4)];
        }
#endif
    }
}

/** Documented at declaration */
void
gpujpeg_dct_gpu(struct gpujpeg_encoder* encoder)
{
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;

    // Encode each component
    for ( int comp = 0; comp < coder->param_image.comp_count; comp++ ) {
        // Get component
        struct gpujpeg_component* component = &coder->component[comp];

        // Determine table type
        enum gpujpeg_component_type type = (comp == 0) ? GPUJPEG_COMPONENT_LUMINANCE : GPUJPEG_COMPONENT_CHROMINANCE;

        int roi_width = component->data_width;
        int roi_height = component->data_height;
        assert(GPUJPEG_BLOCK_SIZE == 8);

        int block_count_x = roi_width / GPUJPEG_BLOCK_SIZE;
        int block_count_y = roi_height / GPUJPEG_BLOCK_SIZE;
        
        // Scales of outputs of 1D DCT.
        const double dct_scales[8] = {1.0, 1.387039845, 1.306562965, 1.175875602, 1.0, 0.785694958, 0.541196100, 0.275899379};
        
        // Prepare quantization table for GPU
        const uint8_t* const raw_quant = encoder->table_quantization[type].table_raw;
        float h_quantization_table[64];
        for( int y = 0; y < 8; y++ ) {
            for( int x = 0; x < 8; x++ ) {
                const int quant_idx = x + 8 * y;
                h_quantization_table[quant_idx] = 1.0 / (raw_quant[quant_idx] * dct_scales[y] * dct_scales[x] * 8); // 8 is the gain of 2D DCT
            }
        }
        
        // Copy quantization table to constant memory
        hipMemcpyToSymbol(HIP_SYMBOL(
            gpujpeg_dct_gpu_quantization_table),
            h_quantization_table, 
            64 * sizeof(*gpujpeg_dct_gpu_quantization_table),
            0,
            hipMemcpyHostToDevice
        );
        gpujpeg_cuda_check_error("Copy DCT quantization table to constant memory");

        enum { WARP_COUNT = 4 };
        
        // Perform block-wise DCT processing
        dim3 dct_grid(
            gpujpeg_div_and_round_up(block_count_x, 4),
            gpujpeg_div_and_round_up(block_count_y, WARP_COUNT),
            1
        );
        dim3 dct_block(4 * 8, WARP_COUNT);
        gpujpeg_dct_gpu_kernel<WARP_COUNT><<<dct_grid, dct_block>>>(
            block_count_x,
            block_count_y,
            component->d_data,
            component->data_width,
            component->d_data_quantized,
            component->data_width * GPUJPEG_BLOCK_SIZE
        );
        hipDeviceSynchronize();
        gpujpeg_cuda_check_error("Forward Integer DCT failed");
    }
}

/** Documented at declaration */
void
gpujpeg_idct_gpu(struct gpujpeg_decoder* decoder)
{
    // Get coder
    struct gpujpeg_coder* coder = &decoder->coder;

    // Encode each component
    for ( int comp = 0; comp < coder->param_image.comp_count; comp++ ) {
        // Get component
        struct gpujpeg_component* component = &coder->component[comp];

        // Determine table type
        enum gpujpeg_component_type type = (comp == 0) ? GPUJPEG_COMPONENT_LUMINANCE : GPUJPEG_COMPONENT_CHROMINANCE;

        int roi_width = component->data_width;
        int roi_height = component->data_height;
        assert(GPUJPEG_BLOCK_SIZE == 8);

        int block_count_x = roi_width / GPUJPEG_BLOCK_SIZE;
        int block_count_y = roi_height / GPUJPEG_BLOCK_SIZE;
        
        // Get quantization table
        uint16_t* d_quantization_table = decoder->table_quantization[type].d_table;
        
        // Copy quantization table to constant memory
        hipMemcpyToSymbol(
            (const char*)gpujpeg_idct_gpu_quantization_table,
            d_quantization_table, 
            64 * sizeof(uint16_t),
            0,
            hipMemcpyDeviceToDevice
        );
        gpujpeg_cuda_check_error("Copy IDCT quantization table to constant memory");

        // Perform block-wise IDCT processing
        dim3 dct_grid(
            gpujpeg_div_and_round_up(block_count_x, GPUJPEG_DCT_BLOCK_COUNT_X),
            gpujpeg_div_and_round_up(block_count_y, GPUJPEG_DCT_BLOCK_COUNT_Y),
            1
        );
        dim3 dct_block(
            GPUJPEG_BLOCK_SIZE,
            GPUJPEG_DCT_BLOCK_COUNT_X,
            GPUJPEG_DCT_BLOCK_COUNT_Y
        );
        gpujpeg_idct_gpu_kernel<<<dct_grid, dct_block>>>(
            block_count_x,
            block_count_y,
            component->d_data_quantized,
            component->data_width * GPUJPEG_BLOCK_SIZE,
            component->d_data,
            component->data_width,
            d_quantization_table
        );
        hipDeviceSynchronize();
        gpujpeg_cuda_check_error("Inverse Integer DCT failed");
    }
}
